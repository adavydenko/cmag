#include "hip/hip_runtime.h"
#include <cstdlib>
#include <cstdio>
#include <string.h>
#include <GL/freeglut.h>
#include "thrust/device_ptr.h"
#include "thrust/for_each.h"
#include "thrust/iterator/zip_iterator.h"
#include "thrust/sort.h"

#include <cuda_gl_interop.h>

#include "helper_timer.h"
#include "hip/hip_runtime_api.h"
#include "helper_cuda_gl.h"

#include "poiseuilleFlowKernel.cu"
extern "C"
{	
	void setParameters(PoiseuilleParams *hostParams){
		checkCudaErrors( hipMemcpyToSymbol(HIP_SYMBOL(params), hostParams, sizeof(PoiseuilleParams)) );
	}

	uint iDivUp(uint a, uint b){
		return (a % b != 0) ? (a / b + 1) : (a / b);
	}

	void computeGridSize(uint n, uint blockSize, uint &numBlocks, uint &numThreads){
		numThreads = min(blockSize, n);
		numBlocks = iDivUp(n, numThreads);
	}

	void cudaGLInit(int argc, char **argv)
	{   
		gpuGLDeviceInit(argc, (const char **)argv); //todo: init
		// if( checkCmdLineFlag(argc, (const char**)argv, "device") ) {
		// 	// cutilDeviceInit(argc, argv);
		// 	gpuDeviceInit(0);
		// } else {
		// 	//cudaGLSetGLDevice( gpuGetMaxGflopsDeviceId() ); //todo: hiden?
		// }
	}

	void registerGLBufferObject(uint vbo, struct hipGraphicsResource **cuda_vbo_resource)
	{
		checkCudaErrors(hipGraphicsGLRegisterBuffer(cuda_vbo_resource, vbo, 
							   cudaGraphicsMapFlagsNone));
	}

	void unregisterGLBufferObject(struct hipGraphicsResource *cuda_vbo_resource)
	{
		checkCudaErrors(hipGraphicsUnregisterResource(cuda_vbo_resource));	
	}

	void *mapGLBufferObject(struct hipGraphicsResource **cuda_vbo_resource)
	{
		void *ptr;
		checkCudaErrors(hipGraphicsMapResources(1, cuda_vbo_resource, 0));
		size_t num_bytes; 
		checkCudaErrors(hipGraphicsResourceGetMappedPointer((void **)&ptr, &num_bytes,  
								   *cuda_vbo_resource));
		return ptr;
	}

	void unmapGLBufferObject(struct hipGraphicsResource *cuda_vbo_resource)
	{
	   checkCudaErrors(hipGraphicsUnmapResources(1, &cuda_vbo_resource, 0));
	}

	void allocateArray(void **devPtr, size_t size)
	{
		checkCudaErrors(hipMalloc(devPtr, size));
	}

	void freeArray(void *devPtr)
	{
		checkCudaErrors(hipFree(devPtr));
	}

	void copyArrayToDevice(void* device, const void* host, int offset, int size)
	{
		checkCudaErrors(hipMemcpy((char *) device + offset, host, size, hipMemcpyHostToDevice));
	}

	void integratePoiseuilleSystem(
		float *pos,
		float *vel,  
		float* velLeapFrog,
		float *acc,
		uint numParticles){
			uint numThreads, numBlocks;
			computeGridSize(numParticles, 256, numBlocks, numThreads);

			integratePoiseuilleSystemD<<< numBlocks, numThreads >>>(
				(float4*)pos,
				(float4*)vel,
				(float4*)velLeapFrog,
				(float4*)acc,
				numParticles);
		    
			//checkCudaErrors("integrate kernel execution failed");
	}

	void sortParticles(uint *dHash, uint *dIndex, uint numParticles)
	{
		thrust::sort_by_key(thrust::device_ptr<uint>(dHash),
							thrust::device_ptr<uint>(dHash + numParticles),
							thrust::device_ptr<uint>(dIndex));
	}

	void calculatePoiseuilleHash(
		uint* gridParticleHash,
		uint* gridParticleIndex,
		float* pos, 
		int numParticles){
			uint numThreads, numBlocks;
			computeGridSize(numParticles, 256, numBlocks, numThreads);

			calculatePoiseuilleHashD<<< numBlocks, numThreads >>>(
				gridParticleHash,
				gridParticleIndex,
				(float4 *) pos,
				numParticles);
		    
			//checkCudaErrors("Kernel execution failed: calculatePoiseuilleHashD");
	}

	void reorderPoiseuilleData(
		uint*  cellStart,
		uint*  cellEnd,
		float* sortedPos,
		float* sortedVel,
		uint*  gridParticleHash,
		uint*  gridParticleIndex,
		float* oldPos,
		float* oldVel,
		uint   numParticles,
		uint   numCells){
			uint numThreads, numBlocks;
			computeGridSize(numParticles, 256, numBlocks, numThreads);

			checkCudaErrors(hipMemset(cellStart, 0xffffffff, numCells*sizeof(uint)));

			#if USE_TEX
				checkCudaErrors(hipBindTexture(0, oldPosTex, oldPos, numParticles*sizeof(float4)));
				checkCudaErrors(hipBindTexture(0, oldVelTex, oldVel, numParticles*sizeof(float4)));
			#endif

				uint smemSize = sizeof(uint)*(numThreads+1);
				reorderPoiseuilleDataD<<< numBlocks, numThreads, smemSize>>>(
					cellStart,
					cellEnd,
					(float4 *) sortedPos,
					(float4 *) sortedVel,
					gridParticleHash,
					gridParticleIndex,
					(float4 *) oldPos,
					(float4 *) oldVel,
					numParticles);
				//checkCudaErrors("Kernel execution failed: reorderPoiseuilleDataD");

			#if USE_TEX
				checkCudaErrors(hipUnbindTexture(oldPosTex));
				checkCudaErrors(hipUnbindTexture(oldVelTex));
			#endif
	}

	void calculatePoiseuilleDensity(			
		float* measures,
		float* sortedPos,	
		float* sortedVel,
		uint* gridParticleIndex,
		uint* cellStart,
		uint* cellEnd,
		uint numParticles,
		uint numGridCells){
			#if USE_TEX
			checkCudaErrors(hipBindTexture(0, oldPosTex, sortedPos, numParticles*sizeof(float4)));
			checkCudaErrors(hipBindTexture(0, oldVelTex, sortedVel, numParticles*sizeof(float4)));
			checkCudaErrors(hipBindTexture(0, cellStartTex, cellStart, numGridCells*sizeof(uint)));
			checkCudaErrors(hipBindTexture(0, cellEndTex, cellEnd, numGridCells*sizeof(uint)));    
			#endif

			uint numThreads, numBlocks;
			computeGridSize(numParticles, 64, numBlocks, numThreads);

			calculatePoiseuilleDensityD<<< numBlocks, numThreads >>>(										  
				(float4*)measures,
				(float4*)sortedPos,  
				(float4*)sortedVel,
				gridParticleIndex,
				cellStart,
				cellEnd,
				numParticles);

			//cutilCheckMsg("Kernel execution failed");
			//checkCudaErrors("Kernel execution failed");

			#if USE_TEX
			checkCudaErrors(hipUnbindTexture(oldPosTex));
			checkCudaErrors(hipUnbindTexture(oldVelTex));
			checkCudaErrors(hipUnbindTexture(cellStartTex));
			checkCudaErrors(hipUnbindTexture(cellEndTex));
			#endif
	}

	void calculatePoiseuilleAcceleration(
		float* acceleration,
		float* sortedMeasures,			
		float* sortedPos,			
		float* sortedVel,
		uint* gridParticleIndex,
		uint* cellStart,
		uint* cellEnd,
		uint numParticles,
		uint numGridCells){
			#if USE_TEX
			checkCudaErrors(hipBindTexture(0, oldPosTex, sortedPos, numParticles*sizeof(float4)));
			checkCudaErrors(hipBindTexture(0, oldVelTex, sortedVel, numParticles*sizeof(float4)));
			checkCudaErrors(hipBindTexture(0, oldMeasuresTex, sortedMeasures, numParticles*sizeof(float4)));
			checkCudaErrors(hipBindTexture(0, cellStartTex, cellStart, numGridCells*sizeof(uint)));
			checkCudaErrors(hipBindTexture(0, cellEndTex, cellEnd, numGridCells*sizeof(uint)));    
			#endif

			uint numThreads, numBlocks;
			computeGridSize(numParticles, 64, numBlocks, numThreads);

			calculatePoiseuilleAccelerationD<<< numBlocks, numThreads >>>(
				(float4*)acceleration,
				(float4*)sortedMeasures,										  
				(float4*)sortedPos,                                          
				(float4*)sortedVel, 
				gridParticleIndex,
				cellStart,
				cellEnd,
				numParticles);

			//cutilCheckMsg("Kernel execution failed");
			//checkCudaErrors("Kernel execution failed");

			#if USE_TEX
			checkCudaErrors(hipUnbindTexture(oldPosTex));
			checkCudaErrors(hipUnbindTexture(oldVelTex));
			checkCudaErrors(hipUnbindTexture(oldMeasuresTex));
			checkCudaErrors(hipUnbindTexture(cellStartTex));
			checkCudaErrors(hipUnbindTexture(cellEndTex));
			#endif
	}
}// extern "C"

