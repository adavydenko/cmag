#include "hip/hip_runtime.h"
#ifndef _PARTICLES_KERNEL_H_
#define _PARTICLES_KERNEL_H_

#include "cutil_math.h"
#include "beam_kernel.cuh"

texture<float4, 1, hipReadModeElementType> oldPosTex;
texture<float4, 1, hipReadModeElementType> oldReferencePosTex;
texture<float4, 1, hipReadModeElementType> olduDisplacementGradientTex;
texture<float4, 1, hipReadModeElementType> oldvDisplacementGradientTex;
texture<float4, 1, hipReadModeElementType> oldwDisplacementGradientTex;
texture<float4, 1, hipReadModeElementType> oldMeasuresTex;
texture<uint, 1, hipReadModeElementType> cellStartTex;
texture<uint, 1, hipReadModeElementType> cellEndTex;

__constant__ SimParams params;


struct Matrix
{
  float a11,a12,a13;
  float a21,a22,a23;
  float a31,a32,a33;
};

__device__ Matrix make_Matrix()
{
  Matrix t; 
  t.a11 = 0; t.a12= 0; t.a13 = 0;
  t.a21 = 0; t.a22= 0; t.a23 = 0;
  t.a31 = 0; t.a32= 0; t.a33 = 0;
  return t;
};
__device__ Matrix operator+ (const Matrix & a, const Matrix & b) 
{ 
	Matrix r;
	r.a11 = a.a11 + b.a11;
	r.a12 = a.a12 + b.a12;
	r.a13 = a.a13 + b.a13;
	r.a21 = a.a21 + b.a21;
	r.a22 = a.a22 + b.a22;
	r.a23 = a.a23 + b.a23;
	r.a31 = a.a31 + b.a31;
	r.a32 = a.a32 + b.a32;
	r.a33 = a.a33 + b.a33;
	return r; 
}
__device__ Matrix operator- (const Matrix & a, const Matrix & b) 
{ 
	Matrix r;
	r.a11 = a.a11 - b.a11;
	r.a12 = a.a12 - b.a12;
	r.a13 = a.a13 - b.a13;
	r.a21 = a.a21 - b.a21;
	r.a22 = a.a22 - b.a22;
	r.a23 = a.a23 - b.a23;
	r.a31 = a.a31 - b.a31;
	r.a32 = a.a32 - b.a32;
	r.a33 = a.a33 - b.a33;
	return r; 
}

__device__ Matrix operator+= (Matrix & a, const Matrix & b) 
{ 	
	a.a11 += b.a11;
	a.a12 += b.a12;
	a.a13 += b.a13;

	a.a21 += b.a21;
	a.a22 += b.a22;
	a.a23 += b.a23;

	a.a31 += b.a31;
	a.a32 += b.a32;
	a.a33 += b.a33;
	return a; 
}

__device__ Matrix operator* (const Matrix & a, const Matrix & b) 
{ 
	Matrix r;
	r.a11 = a.a11 * b.a11 + a.a12 * b.a21 + a.a13 * b.a31;
	r.a12 = a.a11 * b.a12 + a.a12 * b.a22 + a.a13 * b.a32;
	r.a13 = a.a11 * b.a13 + a.a12 * b.a23 + a.a13 * b.a33;

	r.a21 = a.a21 * b.a11 + a.a22 * b.a21 + a.a23 * b.a31;
	r.a22 = a.a21 * b.a12 + a.a22 * b.a22 + a.a23 * b.a32;
	r.a23 = a.a21 * b.a13 + a.a22 * b.a23 + a.a23 * b.a33;

	r.a31 = a.a31 * b.a11 + a.a32 * b.a21 + a.a33 * b.a31;
	r.a32 = a.a31 * b.a12 + a.a32 * b.a22 + a.a33 * b.a32;
	r.a33 = a.a31 * b.a13 + a.a32 * b.a23 + a.a33 * b.a33;
	return r; 
}

__device__ float3 operator* (const Matrix & a, const float3 & b) 
{ 
	float3 r;
	r.x = a.a11 * b.x + a.a12 * b.y + a.a13 * b.z;	
	r.y = a.a21 * b.x + a.a22 * b.y + a.a23 * b.z;	
	r.z = a.a31 * b.x + a.a32 * b.y + a.a33 * b.z;	
	return r; 
}

__device__ Matrix operator* (const float & a, const Matrix & b) 
{ 
	Matrix r;
	r.a11 = a * b.a11;
	r.a12 = a * b.a12;
	r.a13 = a * b.a13;

	r.a21 = a * b.a21;
	r.a22 = a * b.a22;
	r.a23 = a * b.a23;

	r.a31 = a * b.a31;
	r.a32 = a * b.a32;
	r.a33 = a * b.a33;

	return r; 
}

__device__ Matrix Transpose (const Matrix & b) 
{ 
	Matrix r;
	r.a11 = b.a11;
	r.a12 = b.a21;
	r.a13 = b.a31;

	r.a21 = b.a12;
	r.a22 = b.a22;
	r.a23 = b.a32;

	r.a31 = b.a13;
	r.a32 = b.a23;
	r.a33 = b.a33;
	return r; 
}

__device__ int3 calcGridPos(float3 p)
{
    int3 gridPos;
    gridPos.x = floor((p.x - params.worldOrigin.x) / params.cellSize.x);
    gridPos.y = floor((p.y - params.worldOrigin.y) / params.cellSize.y);
    gridPos.z = floor((p.z - params.worldOrigin.z) / params.cellSize.z);
    return gridPos;
}

__device__ uint calcGridHash(int3 gridPos)
{
    gridPos.x = gridPos.x & (params.gridSize.x-1);  
    gridPos.y = gridPos.y & (params.gridSize.y-1);
    gridPos.z = gridPos.z & (params.gridSize.z-1);        
    return __umul24(__umul24(gridPos.z, params.gridSize.y), params.gridSize.x) + __umul24(gridPos.y, params.gridSize.x) + gridPos.x;
}

__global__ void calcHashD(uint* Hash,  // output
               uint* Index, // output
               float4* pos, // input
               uint    numParticles)
{
    uint index = __umul24(blockIdx.x, blockDim.x) + threadIdx.x;
    if (index >= numParticles) return;
    
    volatile float4 p = pos[index];

    int3 gridPos = calcGridPos(make_float3(p.x, p.y, p.z));
    uint hash = calcGridHash(gridPos);

    Hash[index] = hash;
    Index[index] = index;
}

__global__ void reorderDataAndFindCellStartD(
								  uint*   cellStart,// output: 
							      uint*   cellEnd,             // output: 
								  float4* sortedPos,		   // output;
  							      float4* sortedReferencePos,  // output:						  
                                  uint *  Hash,				   // input: 
                                  uint *  Index,			   // input: 
								  float4* oldPos,		   // input;
								  float4* oldReferencePos,
							      uint    numParticles)
{
	extern __shared__ uint sharedHash[];    
    uint index = __umul24(blockIdx.x,blockDim.x) + threadIdx.x;
	
    volatile uint hash;   
    if (index < numParticles) {
         hash = Hash[index];
        
	    sharedHash[threadIdx.x+1] = hash;

	    if (index > 0 && threadIdx.x == 0)
	    {		    
		    sharedHash[0] = Hash[index-1];
	    }
	}

	__syncthreads();
	
	if (index < numParticles) {		
	    if (index == 0 || hash != sharedHash[threadIdx.x])
	    {
		    cellStart[hash] = index;
            if (index > 0)
                cellEnd[sharedHash[threadIdx.x]] = index;
	    }

        if (index == numParticles - 1)
        {
            cellEnd[hash] = index + 1;
        }

	    uint sortedIndex = Index[index];		 				
		sortedPos[index] = FETCH(oldPos, sortedIndex);
        sortedReferencePos[index] = FETCH(oldReferencePos, sortedIndex);
	}
}


__device__ float sumDensityPart(
				   int3    gridPos,
                   uint    index,
                   float3  referencePos,
                   float4* oldReferencePos, 
                   uint*   cellStart,
                   uint*   cellEnd)
{
    uint gridHash = calcGridHash(gridPos);

    uint startIndex = FETCH(cellStart, gridHash);

    float sum = 0.0f;
    if (startIndex != 0xffffffff) {                
        uint endIndex = FETCH(cellEnd, gridHash);
        for(uint j=startIndex; j<endIndex; j++) {
            if (j != index) {             
	            float3 referencePos_j = make_float3(FETCH(oldReferencePos, j));
				float wpolyExpr = 0.0f;

				float3 relPos = referencePos_j - referencePos; 
				float dist = length(relPos);

				if (dist < params.smoothingRadius) {					
					wpolyExpr = params.c * 2 * params.smoothingRadius / HIP_PI_F;					
					sum += wpolyExpr * cos( 0.5 * (dist + params.smoothingRadius) * HIP_PI_F / params.smoothingRadius) + wpolyExpr;
				}                
            }
        }
    }
    return sum;
}

__global__ 
void calcDensityD(			
			float4* measures, //output
			float4* oldReferencePos,	 //input sorted position					
			uint* cellStart,
			uint* cellEnd,
			uint numParticles)
			
{
	uint index = __mul24(blockIdx.x,blockDim.x) + threadIdx.x;
    if (index >= numParticles) return;    

	float3 referencePos = make_float3(FETCH(oldReferencePos, index));
    int3 gridPos = calcGridPos(referencePos);

    float sum = 0.0f;
	int cellcount = 1;
    for(int z=-cellcount; z<=cellcount; z++) {
        for(int y=-cellcount; y<=cellcount; y++) {
            for(int x=-cellcount; x<=cellcount; x++) {
                int3 neighbourPos = gridPos + make_int3(x, y, z);
                sum += sumDensityPart(neighbourPos, index, referencePos, oldReferencePos, cellStart, cellEnd);
            }
        }
    }	
	float dens = sum * params.particleMass + params.restDensity;
    measures[index].x = dens;	//density	
	measures[index].y = params.particleMass / dens;	//volume
}

__device__ Matrix sumDisplacementGradientPart(
				   int3    gridPos,
                   uint    index,
                   float3  pos_i,
                   float4* oldPos, 
				   float3  referencePos_i,
                   float4* oldReferencePos, 
				   float4* oldMeasures,
                   uint*   cellStart,
                   uint*   cellEnd)
{
	uint gridHash = calcGridHash(gridPos);

    uint startIndex = FETCH(cellStart, gridHash);    	
	Matrix gradient = make_Matrix();	
	
    if (startIndex != 0xffffffff) {               
        uint endIndex = FETCH(cellEnd, gridHash);
        for(uint j=startIndex; j<endIndex; j++) {
            if (j != index) {             
	            float3 pos_j = make_float3(FETCH(oldPos, j));				
				float3 referencePos_j = make_float3(FETCH(oldReferencePos,j));
				float volume_j = FETCH(oldMeasures, j).y;

				float3 relPos = referencePos_i - referencePos_j;
				float dist = length(relPos);
				

				if (dist < params.smoothingRadius) {				
					float tempExpr =  sinf((dist + params.smoothingRadius) * HIP_PI_F / (2.0f * params.smoothingRadius) );			
					gradient.a11 += volume_j * (pos_j.x - pos_i.x - (referencePos_j.x - referencePos_i.x)) * params.SpikyKern * tempExpr * (relPos.x / dist);
					gradient.a12 += volume_j * (pos_j.x - pos_i.x - (referencePos_j.x - referencePos_i.x)) * params.SpikyKern * tempExpr * (relPos.y / dist);
					gradient.a13 += volume_j * (pos_j.x - pos_i.x - (referencePos_j.x - referencePos_i.x)) * params.SpikyKern * tempExpr * (relPos.z / dist);
					
					gradient.a21 += volume_j * (pos_j.y - pos_i.y - (referencePos_j.y - referencePos_i.y)) * params.SpikyKern * tempExpr * (relPos.x / dist);
					gradient.a22 += volume_j * (pos_j.y - pos_i.y - (referencePos_j.y - referencePos_i.y)) * params.SpikyKern * tempExpr * (relPos.y / dist);
					gradient.a23 += volume_j * (pos_j.y - pos_i.y - (referencePos_j.y - referencePos_i.y)) * params.SpikyKern * tempExpr * (relPos.z / dist);

					gradient.a31 += volume_j * (pos_j.z - pos_i.z - (referencePos_j.z - referencePos_i.z)) * params.SpikyKern * tempExpr * (relPos.x / dist);
					gradient.a32 += volume_j * (pos_j.z - pos_i.z - (referencePos_j.z - referencePos_i.z)) * params.SpikyKern * tempExpr * (relPos.y / dist);
					gradient.a33 += volume_j * (pos_j.z - pos_i.z - (referencePos_j.z - referencePos_i.z)) * params.SpikyKern * tempExpr * (relPos.z / dist);																				
				}                
            }
        }
    }
	return gradient;		
}

__global__ void calcDisplacementGradientD(
						  float4* udisplacementGradient,
						  float4* vdisplacementGradient,
						  float4* wdisplacementGradient,
						  float4* oldPos,	
						  float4* oldReferencePos,	
						  float4* oldMeasures,
						  uint* Index, 
						  uint* cellStart,
						  uint* cellEnd,
						  uint numParticles)			
{
	uint index = __mul24(blockIdx.x,blockDim.x) + threadIdx.x;
    if (index >= numParticles) return;    

	float3 pos = make_float3(FETCH(oldPos, index));
	float3 referencePos = make_float3(FETCH(oldReferencePos, index));
    int3 gridPos = calcGridPos(referencePos);	
	Matrix buf = make_Matrix();	
	int cellcount = 1;
    for(int z=-cellcount; z<=cellcount; z++) {
        for(int y=-cellcount; y<=cellcount; y++) {
            for(int x=-cellcount; x<=cellcount; x++) {
                int3 neighbourPos = gridPos + make_int3(x, y, z);
                buf += sumDisplacementGradientPart(neighbourPos, index, pos, oldPos, referencePos, oldReferencePos, oldMeasures, cellStart, cellEnd);				
            }
        }
    }    				
	udisplacementGradient[index].x = buf.a11;
	udisplacementGradient[index].y = buf.a12;
	udisplacementGradient[index].z = buf.a13;

	vdisplacementGradient[index].x = buf.a21;
	vdisplacementGradient[index].y = buf.a22;
	vdisplacementGradient[index].z = buf.a23;

	wdisplacementGradient[index].x = buf.a31;
	wdisplacementGradient[index].y = buf.a32;	
	wdisplacementGradient[index].z = buf.a33;
}

__device__ float3 sumForcePart(
				   int3    gridPos,
                   uint    index,
                   float3  referencePos_i,
                   float4* oldReferencePos, 
				   float3  du_i,
				   float3  dv_i,
				   float3  dw_i,
                   float   volume_i, 
				   float4* oldMeasures,
                   uint*   cellStart,
                   uint*   cellEnd)
{
	uint gridHash = calcGridHash(gridPos);
    uint startIndex = FETCH(cellStart, gridHash);    
	float3 tmpForce = make_float3(0.0f);	
	Matrix Sigma = make_Matrix();
	float3 d = make_float3(0.0f);			
	Matrix I = make_Matrix();
	I.a11 = 1; I.a22 = 1; I.a33 = 1;		
	Matrix dUT = make_Matrix();
	Matrix J = make_Matrix();	
	Matrix E = make_Matrix();	

	dUT.a11 = du_i.x;
	dUT.a12 = du_i.y;
	dUT.a13 = du_i.z;

	dUT.a21 = dv_i.x;
	dUT.a22 = dv_i.y;
	dUT.a23 = dv_i.z;

	dUT.a31 = dw_i.x;
	dUT.a32 = dw_i.y;
	dUT.a33 = dw_i.z;

	J = I + dUT;				
	
	//Green-Saint-Venant strain tensor	
	E = 0.5 * ((Transpose(J)*J) - I);	
	//E.a22 = 0.5 * ( (1+dUT.a22)*(1+dUT.a22) - 1);	

	float t1 = E.a11;
	float t2 = E.a22;
	float t3 = E.a33;

	//Stress tensor
	Sigma.a11 = (params.Young / ( 1 + params.Poisson))*(t1 + (params.Poisson / ( 1 - 2 * params.Poisson))*(E.a11 + E.a22 + E.a33));
	Sigma.a22 = (params.Young / ( 1 + params.Poisson))*(t2 + (params.Poisson / ( 1 - 2 * params.Poisson))*(E.a11 + E.a22 + E.a33));
	Sigma.a33 = (params.Young / ( 1 + params.Poisson))*(t3 + (params.Poisson / ( 1 - 2 * params.Poisson))*(E.a11 + E.a22 + E.a33));
	
	Sigma.a12 = Sigma.a21 = (params.Young / (1 + params.Poisson))*E.a12;
	Sigma.a13 = Sigma.a31 = (params.Young / (1 + params.Poisson))*E.a13;
	Sigma.a23 = Sigma.a32 = (params.Young / (1 + params.Poisson))*E.a23;		

    if (startIndex != 0xffffffff) {               
        uint endIndex = FETCH(cellEnd, gridHash);
        for(uint j=startIndex; j<endIndex; j++) {
            if (j != index) {             
	            float3 referencePos_j = make_float3(FETCH(oldReferencePos, j));
				float4 measure = FETCH(oldMeasures, j);				
				float volume_j = measure.y;
				float tempExpr = 0.0f;
				float3 relPos = referencePos_i - referencePos_j;

				float dist = length(relPos);
				if (dist < params.smoothingRadius) {					
					tempExpr = sinf((dist + params.smoothingRadius) * HIP_PI_F / (2.0f * params.smoothingRadius) );				
					d.x = volume_j * params.SpikyKern * (relPos.x / dist) * tempExpr;
					d.y = volume_j * params.SpikyKern * (relPos.y / dist) * tempExpr;
					d.z = volume_j * params.SpikyKern * (relPos.z / dist) * tempExpr;																		

					tmpForce += -volume_i * (((I + dUT) * Sigma) * d);
					//tmpForce.y += -volume_i * ( (1 + dUT.a22) * Sigma.a22 * d.y );
				}                
            }
        }
    }
	return tmpForce;
}

__global__ void calcAccelerationD(
						  float4* acceleration,
						  float4* oldPos,	
						  float4* oldReferencePos,	
						  float4* olduDisplacementGradient,	
						  float4* oldvDisplacementGradient,	
						  float4* oldwDisplacementGradient,	
						  float4* oldMeasures,
						  uint* Index, 
						  uint* cellStart,
						  uint* cellEnd,
						  uint numParticles)			
{
	uint index = __mul24(blockIdx.x,blockDim.x) + threadIdx.x;
    if (index >= numParticles) return;    

	float3 pos = make_float3(FETCH(oldPos, index));
	float3 referencePos = make_float3(FETCH(oldReferencePos, index));
	float3 du_i = make_float3(FETCH(olduDisplacementGradient, index));
	float3 dv_i = make_float3(FETCH(oldvDisplacementGradient, index));
	float3 dw_i = make_float3(FETCH(oldwDisplacementGradient, index));
	float volume_i = FETCH(oldMeasures, index).y;

    int3 gridPos = calcGridPos(referencePos);
	float3 force = make_float3(0.0f);
	int cellcount = 1;
    for(int z=-cellcount; z<=cellcount; z++) {
        for(int y=-cellcount; y<=cellcount; y++) {
            for(int x=-cellcount; x<=cellcount; x++) {
                int3 neighbourPos = gridPos + make_int3(x, y, z);
				//!!! -=
                force -= sumForcePart(neighbourPos, index, referencePos, oldReferencePos, du_i, dv_i, dw_i, volume_i, oldMeasures, cellStart, cellEnd);
            }
        }
    }    	
	uint originalIndex = Index[index];
	float3 acc = force /  params.particleMass;//make_float3(0,0,0);//
	acceleration[originalIndex] =  make_float4(acc, 0.0f);
}

__global__ void integrate(float4* posArray, //input / output 
						  float4* velArray, //input / output
						  float4* accArray, //input
						  uint numParticles)
{
    uint index = __umul24(blockIdx.x,blockDim.x) + threadIdx.x;
    if (index >= numParticles) return;

	volatile float4 posData = posArray[index];
	volatile float4 velData = velArray[index];	
	volatile float4 accData = accArray[index];	
	
	float3 pos = make_float3(posData.x, posData.y, posData.z);
    float3 vel = make_float3(velData.x, velData.y, velData.z);
	float3 acc = make_float3(accData.x, accData.y, accData.z);

	vel += (params.gravity + acc) * params.deltaTime * velData.w;
    pos += vel * params.deltaTime;  

	posArray[index] = make_float4(pos, posData.w);
	velArray[index] = make_float4(vel, velData.w);
	//accArray[index] = make_float4(0.0f);
}
#endif
