#include "hip/hip_runtime.h"
#include <cutil_inline.h>
#include <cstdlib>
#include <cstdio>
#include <string.h>
#include <GL/glut.h>
#include <cuda_gl_interop.h>
#include "fluid_kernel.cu"
#include "magUtil.cuh"
extern "C"
{	
	void setParameters(SimParams *hostParams){
		cutilSafeCall( hipMemcpyToSymbol(HIP_SYMBOL(params), hostParams, sizeof(SimParams)) );
	}	

	void removeRightBoundary(
		float * position,
		uint numParticles){
			uint numThreads, numBlocks;
			computeGridSize(numParticles, 256, numBlocks, numThreads);

			removeRightBoundaryD<<< numBlocks, numThreads >>>(
				(float4*)position,
				numParticles);
		    
			cutilCheckMsg("removeRightBoundary kernel execution failed");
	}

	void integrateSystem(
		float *pos,
		float *vel,  
		float* velLeapFrog,
		float *acc,
		uint numParticles){
			uint numThreads, numBlocks;
			computeGridSize(numParticles, 256, numBlocks, numThreads);

			integrate<<< numBlocks, numThreads >>>(
				(float4*)pos,
				(float4*)vel,
				(float4*)velLeapFrog,
				(float4*)acc,
				numParticles);
		    
			cutilCheckMsg("integrate kernel execution failed");
	}

	void calcHash(
		uint* gridParticleHash,
		uint* gridParticleIndex,
		float* pos, 
		int numParticles){
			uint numThreads, numBlocks;
			computeGridSize(numParticles, 256, numBlocks, numThreads);

			calcHashD<<< numBlocks, numThreads >>>(
				gridParticleHash,
				gridParticleIndex,
				(float4 *) pos,
				numParticles);
		    
			cutilCheckMsg("Kernel execution failed");
	}

	void reorderDataAndFindCellStart(
		uint*  cellStart,
		uint*  cellEnd,
		float* sortedPos,
		float* sortedVel,
		uint*  gridParticleHash,
		uint*  gridParticleIndex,
		float* oldPos,
		float* oldVel,
		uint   numParticles,
		uint   numCells){
			uint numThreads, numBlocks;
			computeGridSize(numParticles, 256, numBlocks, numThreads);

			cutilSafeCall(hipMemset(cellStart, 0xffffffff, numCells*sizeof(uint)));

			#if USE_TEX
				cutilSafeCall(hipBindTexture(0, oldPosTex, oldPos, numParticles*sizeof(float4)));
				cutilSafeCall(hipBindTexture(0, oldVelTex, oldVel, numParticles*sizeof(float4)));
			#endif

				uint smemSize = sizeof(uint)*(numThreads+1);
				reorderDataAndFindCellStartD<<< numBlocks, numThreads, smemSize>>>(
					cellStart,
					cellEnd,
					(float4 *) sortedPos,
					(float4 *) sortedVel,
					gridParticleHash,
					gridParticleIndex,
					(float4 *) oldPos,
					(float4 *) oldVel,
					numParticles);
				cutilCheckMsg("Kernel execution failed: reorderDataAndFindCellStartD");

			#if USE_TEX
				cutilSafeCall(hipUnbindTexture(oldPosTex));
				cutilSafeCall(hipUnbindTexture(oldVelTex));
			#endif
	}

	void calculateDensityVariation(			
		float* sortedVariations,
		float* sortedMeasures,
		float* sortedPos,			
		float* sortedVel,		
		uint* gridParticleIndex,
		uint* cellStart,
		uint* cellEnd,
		uint numParticles,
		uint numGridCells){
			#if USE_TEX
			cutilSafeCall(hipBindTexture(0, oldPosTex, sortedPos, numParticles*sizeof(float4)));
			cutilSafeCall(hipBindTexture(0, oldMeasuresTex, sortedMeasures, numParticles*sizeof(float4)));
			cutilSafeCall(hipBindTexture(0, oldVelTex, sortedVel, numParticles*sizeof(float4)));
			cutilSafeCall(hipBindTexture(0, cellStartTex, cellStart, numGridCells*sizeof(uint)));
			cutilSafeCall(hipBindTexture(0, cellEndTex, cellEnd, numGridCells*sizeof(uint)));    
			#endif

			uint numThreads, numBlocks;
			computeGridSize(numParticles, 64, numBlocks, numThreads);

			calculateDensityVariationD<<< numBlocks, numThreads >>>(										  
				(float4*)sortedVariations,
				(float4*)sortedMeasures,
				(float4*)sortedPos,                                          
				(float4*)sortedVel, 
				gridParticleIndex,
				cellStart,
				cellEnd,
				numParticles);

			cutilCheckMsg("Kernel execution failed");

			#if USE_TEX
			cutilSafeCall(hipUnbindTexture(oldPosTex));
			cutilSafeCall(hipUnbindTexture(oldMeasuresTex));
			cutilSafeCall(hipUnbindTexture(oldVelTex));
			cutilSafeCall(hipUnbindTexture(cellStartTex));
			cutilSafeCall(hipUnbindTexture(cellEndTex));			
			#endif
	}

	void calculateDensity(			
		float* sortedMeasures,		
		float* sortedVariations,	
		uint numParticles,
		uint numGridCells){
			#if USE_TEX
			cutilSafeCall(hipBindTexture(0, oldVariationsTex, sortedVariations, numParticles*sizeof(float4)));
			#endif
			uint numThreads, numBlocks;
			computeGridSize(numParticles, 64, numBlocks, numThreads);

			calculateDensityD<<< numBlocks, numThreads >>>(										  
				(float4*)sortedMeasures,
				(float4*)sortedVariations,	
				numParticles);

			cutilCheckMsg("Kernel execution failed");

			#if USE_TEX
			cutilSafeCall(hipUnbindTexture(oldVariationsTex));
			#endif
	}

	void calcAndApplyAcceleration(
		float* acceleration,
		float* sortedMeasures,			
		float* sortedPos,			
		float* sortedVel,
		uint* gridParticleIndex,
		uint* cellStart,
		uint* cellEnd,
		uint numParticles,
		uint numGridCells){
			#if USE_TEX
			cutilSafeCall(hipBindTexture(0, oldPosTex, sortedPos, numParticles*sizeof(float4)));
			cutilSafeCall(hipBindTexture(0, oldVelTex, sortedVel, numParticles*sizeof(float4)));
			cutilSafeCall(hipBindTexture(0, oldMeasuresTex, sortedMeasures, numParticles*sizeof(float4)));
			cutilSafeCall(hipBindTexture(0, cellStartTex, cellStart, numGridCells*sizeof(uint)));
			cutilSafeCall(hipBindTexture(0, cellEndTex, cellEnd, numGridCells*sizeof(uint)));    
			#endif

			uint numThreads, numBlocks;
			computeGridSize(numParticles, 64, numBlocks, numThreads);

			calcAndApplyAccelerationD<<< numBlocks, numThreads >>>(
				(float4*)acceleration,
				(float4*)sortedMeasures,										  
				(float4*)sortedPos,                                          
				(float4*)sortedVel, 
				gridParticleIndex,
				cellStart,
				cellEnd,
				numParticles);

			cutilCheckMsg("Kernel execution failed");

			#if USE_TEX
			cutilSafeCall(hipUnbindTexture(oldPosTex));
			cutilSafeCall(hipUnbindTexture(oldVelTex));
			cutilSafeCall(hipUnbindTexture(oldMeasuresTex));
			cutilSafeCall(hipUnbindTexture(cellStartTex));
			cutilSafeCall(hipUnbindTexture(cellEndTex));
			#endif
	}
}// extern "C"

