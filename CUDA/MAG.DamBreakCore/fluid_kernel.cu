#include "hip/hip_runtime.h"
#include <stdio.h>
#include <math.h>
#include "cutil_math.h"
#include "hip/hip_math_constants.h"
#include "fluid_kernel.cuh"

#if USE_TEX
texture<float4, 1, hipReadModeElementType> oldPosTex;
texture<float4, 1, hipReadModeElementType> oldVelTex;
texture<float4, 1, hipReadModeElementType> oldMeasuresTex;
texture<float4, 1, hipReadModeElementType> oldVariationsTex;

texture<uint, 1, hipReadModeElementType> gridParticleHashTex;
texture<uint, 1, hipReadModeElementType> cellStartTex;
texture<uint, 1, hipReadModeElementType> cellEndTex;
#endif
__constant__ SimParams params;

__device__ int3 calcGridPos(float3 p){
	int3 gridPos;
	gridPos.x = floor((p.x - params.worldOrigin.x) / params.cellSize.x);
	gridPos.y = floor((p.y - params.worldOrigin.y) / params.cellSize.y);
	gridPos.z = floor((p.z - params.worldOrigin.z) / params.cellSize.z);
	return gridPos;
}

__device__ uint calcGridHash(int3 gridPos){
	gridPos.x = gridPos.x & (params.gridSize.x-1);  
	gridPos.y = gridPos.y & (params.gridSize.y-1);
	gridPos.z = gridPos.z & (params.gridSize.z-1);        
	return __umul24(__umul24(gridPos.z, params.gridSize.y), params.gridSize.x) + __umul24(gridPos.y, params.gridSize.x) + gridPos.x;
}

__global__ void calcHashD(
	uint*   gridParticleHash,  // output
	uint*   gridParticleIndex, // output
	float4* pos,               // input
	uint    numParticles){
		uint index = __umul24(blockIdx.x, blockDim.x) + threadIdx.x;
		if (index >= numParticles) return;			    
		volatile float4 p = pos[index];		

		int3 gridPos = calcGridPos(make_float3(p.x, p.y, p.z));
		uint hash = calcGridHash(gridPos);

		gridParticleHash[index] = hash;
		gridParticleIndex[index] = index;
}

__global__ void reorderDataAndFindCellStartD(
	uint*   cellStart,        // output
	uint*   cellEnd,          // output
	float4* sortedPos,        // output
	float4* sortedVel,        // output
	uint *  gridParticleHash, // input
	uint *  gridParticleIndex,// input
	float4* oldPos,           // input
	float4* oldVel,           // input
	uint    numParticles){
		extern __shared__ uint sharedHash[];    // blockSize + 1 elements
		uint index = __umul24(blockIdx.x,blockDim.x) + threadIdx.x;
		
		uint hash;
		if (index < numParticles) {
			hash = gridParticleHash[index];

			sharedHash[threadIdx.x+1] = hash;

			if (index > 0 && threadIdx.x == 0)
			{
				sharedHash[0] = gridParticleHash[index-1];
			}
		}

		__syncthreads();
		
		if (index < numParticles) {
			if (index == 0 || hash != sharedHash[threadIdx.x])
			{
				cellStart[hash] = index;
				if (index > 0)
					cellEnd[sharedHash[threadIdx.x]] = index;
			}

			if (index == numParticles - 1)
			{
				cellEnd[hash] = index + 1;
			}

			uint sortedIndex = gridParticleIndex[index];
			float4 pos = FETCH(oldPos, sortedIndex);       
			float4 vel = FETCH(oldVel, sortedIndex);       

			sortedPos[index] = pos;
			sortedVel[index] = vel;
		}
}

__device__ float sumDensityVariation(
	int3    gridPos,
	uint    index,
	float3  pos,
	float4* oldPos,
	float3  vel,
	float4* oldVel,
	float4* oldMeasures,
	uint*   cellStart,
	uint*   cellEnd){
		uint gridHash = calcGridHash(gridPos);
		uint startIndex = FETCH(cellStart, gridHash);

		float sum = 0.0f;
		if (startIndex != 0xffffffff) {        // cell is not empty
			uint endIndex = FETCH(cellEnd, gridHash);
			for(uint j=startIndex; j<endIndex; j++) {
				if (j != index) {             
					float4 post = FETCH(oldPos, j);
					if(post.w < RightSecondType) // RightFirstType + FirstType
						continue;
					float3 pos2 = make_float3(post);

					//float3 pos2 = make_float3(FETCH(oldPos, j));
					float3 vel2 = make_float3(FETCH(oldVel, j));
					float density2 =FETCH(oldMeasures, j).x;					

					float3 relPos = pos2 - pos; 
					float dist = length(relPos);
					float q = dist / params.smoothingRadius;		

					//float coeff = 7.0f / 4 / HIP_PI_F / powf(params.smoothingRadius, 2);
					//coeff *(powf(1 - 0.5f * q, 4) * (2 * q + 1));	
					float temp = 0.0f;
					float coeff = 7.0f / 2 / HIP_PI_F / powf(params.smoothingRadius, 3);
					if(q < 2){
						temp = coeff * (-powf(1 - 0.5f * q,3) * (2 * q + 1) +powf(1 - 0.5f * q, 4));
						sum += 1.0f / density2 * dot(vel2 - vel, normalize(relPos)) * temp;																					
					}
				}
			}
		}
		return sum;
}

__global__ void calculateDensityVariationD(			
	float4* variations, //output
	float4* oldMeasures, //input
	float4* oldPos,	  //input 
	float4* oldVel,   //input
	uint* gridParticleIndex,
	uint* cellStart,
	uint* cellEnd,
	uint numParticles){
		uint index = __mul24(blockIdx.x,blockDim.x) + threadIdx.x;
		if (index >= numParticles) return;    

		float4 pos1 = FETCH(oldPos, index);
		if(pos1.w < RightSecondType) 
			return;
		float3 pos = make_float3(pos1);

		//float3 pos = make_float3(FETCH(oldPos, index));	
		float3 vel = make_float3(FETCH(oldVel, index));
		int3 gridPos = calcGridPos(pos);

		float sum = 0.0f;		
		for(int z=-params.cellcount; z<=params.cellcount; z++) {
			for(int y=-params.cellcount; y<=params.cellcount; y++) {
				for(int x=-params.cellcount; x<=params.cellcount; x++) {
					int3 neighbourPos = gridPos + make_int3(x, y, z);
					sum += sumDensityVariation(
						neighbourPos,
						index,
						pos,
						oldPos,
						vel,
						oldVel,
						oldMeasures,
						cellStart,
						cellEnd);
				}
			}
		}					
		variations[index].x =  FETCH(oldMeasures, index).x;			
		variations[index].w =  params.particleMass * sum;			
}

__global__ void calculateDensityD(			
	float4* measures, //output	
	float4* oldVariations, //input	
	uint numParticles){
		uint index = __mul24(blockIdx.x,blockDim.x) + threadIdx.x;
		if (index >= numParticles) return;    						

		float oldDens = FETCH(oldVariations, index).x;
		float densVar = FETCH(oldVariations, index).w;
		float newDens = oldDens * (1 + densVar * params.deltaTime);

		measures[index].x = newDens;		
		measures[index].y = params.B * (powf(newDens / params.restDensity ,params.gamma) - 1.0f); 			
}

//__device__ float4 getVelocityDiff(
//	float4 iVelocity, 
//	float4 iPosition, 
//	float4 jVelocity,
//	float4 jPosition)
//{	
//	return iVelocity - jVelocity;
//}

__device__ float3 sumNavierStokesForces(
	int3    gridPos,
	uint    index,
	float3  pos,
	float4* oldPos, 
	float3  vel,
	float4* oldVel,
	float density,
	float pressure,				   
	float4* oldMeasures,
	uint*   cellStart,
	uint*   cellEnd){
		uint gridHash = calcGridHash(gridPos);
		uint startIndex = FETCH(cellStart, gridHash);
	    
		float3 tmpForce = make_float3(0.0f);
		float texp = 0.0f;
		float pexp = 0.0f;
		if (startIndex != 0xffffffff) {               
			uint endIndex = FETCH(cellEnd, gridHash);
			for(uint j=startIndex; j<endIndex; j++) {
				if (j != index) {             

					float4 post = FETCH(oldPos, j);
					float3 pos2 = make_float3(post);
					if(post.w  < RightSecondType) 
					{
						float3 relPos = pos - pos2;
						float dist = length(relPos);
						if(params.a / dist <= 1.0f)
						{							
							tmpForce += params.D * (powf(params.a / dist, 12)
								- powf(params.a / dist, 6)) * relPos / powf(dist, 2);
						}
						continue;
					}					

					//float3 pos2 = make_float3(FETCH(oldPos, j));
					float3 vel2 = make_float3(FETCH(oldVel, j));				
					float4 measure = FETCH(oldMeasures, j);
					float density2 = measure.x;
					float pressure2 = measure.y;				
					float tempExpr = 0.0f;

					float3 relPos = pos - pos2;
					float dist = length(relPos);				

					float q = dist / params.smoothingRadius;		
					float temp = 0.0f;
					float coeff = 7.0f / 2 / HIP_PI_F / powf(params.smoothingRadius, 3);
					if(q < 2){
						temp = coeff * (-powf(1 - 0.5f * q,3) * (2 * q + 1) +powf(1 - 0.5f * q, 4));
						float artViscosity = 0.0f;
						float vij_pij = dot((vel - vel2),relPos);
						
						if(vij_pij < 0){						
							float nu = 2.0f * 0.18f * params.smoothingRadius *
								params.soundspeed / (density + density2);

							artViscosity = -1.0f * nu * vij_pij / 
								(dot(relPos, relPos) + 0.001f * pow(params.smoothingRadius, 2));
						}
						tmpForce +=  -1.0f * params.particleMass *
							(pressure / pow(density,2) + pressure2 / pow(density2,2) +
							artViscosity) * normalize(relPos) * temp;						
					}        
				}
			}
		}
		return tmpForce;				
}

__global__ void calcAndApplyAccelerationD(
	float4* acceleration,			
	float4* oldMeasures,
	float4* oldPos,			
	float4* oldVel,
	uint* gridParticleIndex,
	uint* cellStart,
	uint* cellEnd,
	uint numParticles){
		uint index = __mul24(blockIdx.x,blockDim.x) + threadIdx.x;
		if (index >= numParticles) return;    

		float4 pos1 = FETCH(oldPos, index);
		if(pos1.w  < RightSecondType) 
			return;
		float3 pos = make_float3(pos1);

		//float3 pos = make_float3(FETCH(oldPos, index));
		float3 vel = make_float3(FETCH(oldVel, index));
		float4 measure = FETCH(oldMeasures,index);
		float density = measure.x;
		float pressure = measure.y;

		int3 gridPos = calcGridPos(pos);

		float3 force = make_float3(0.0f);	
		for(int z=-params.cellcount; z<=params.cellcount; z++) {
			for(int y=-params.cellcount; y<=params.cellcount; y++) {
				for(int x=-params.cellcount; x<=params.cellcount; x++) {
					int3 neighbourPos = gridPos + make_int3(x, y, z);
					force += sumNavierStokesForces(neighbourPos, 
						index, 
						pos, 
						oldPos,
						vel,
						oldVel,
						density,
						pressure,					
						oldMeasures,
						cellStart, 
						cellEnd);
				}
			}
		}
		uint originalIndex = gridParticleIndex[index];					
		float3 acc = force;			
		acceleration[originalIndex] =  make_float4(acc, 0.0f);
}

__global__ void removeRightBoundaryD(
	float4* posArray,		 
	uint numParticles){
		uint index = __umul24(blockIdx.x,blockDim.x) + threadIdx.x;
		if (index >= numParticles) return;          		
		
		volatile float4 posData = posArray[index]; 
		if((posData.w != RightFirstType) && (posData.w != RightSecondType))// it's not a right boundary particle
			return;

		float halfWorldXSize = params.gridSize.x * params.particleRadius;		
		/*float halfWorldYSize = params.gridSize.y * params.particleRadius;	
		float halfWorldZSize = params.gridSize.z * params.particleRadius;	*/

		posArray[index] = make_float4(posData.x +halfWorldXSize, posData.y, posData.z, posData.w);
}


__global__ void integrate(
	float4* posArray,		 // input, output
	float4* velArray,		 // input, output  
	float4* velLeapFrogArray, // output
	float4* acceleration,	 // input
	uint numParticles){
		uint index = __umul24(blockIdx.x,blockDim.x) + threadIdx.x;
		if (index >= numParticles) return;          		

		volatile float4 posData = posArray[index]; 
		volatile float4 velData = velArray[index];
		volatile float4 accData = acceleration[index];
		volatile float4 velLeapFrogData = velLeapFrogArray[index];

		if(posData.w != Fluid) //it's not a fluid particle
			return;

		float3 pos = make_float3(posData.x, posData.y, posData.z);
		float3 vel = make_float3(velData.x, velData.y, velData.z);
		float3 acc = make_float3(accData.x, accData.y, accData.z);

		float3 nextVel = vel + (params.gravity + acc) * params.deltaTime;

		float3 velLeapFrog = vel + nextVel;
		velLeapFrog *= 0.5;

		vel = nextVel;   	
		pos += vel * params.deltaTime;   

		float scale = params.gridSize.x * params.particleRadius;
		float bound = 2.0f * params.particleRadius * params.fluidParticlesSize.z - 1.0f * scale;						

		float halfWorldXSize = params.gridSize.x * params.particleRadius;		
		float halfWorldYSize = params.gridSize.y * params.particleRadius;				
	    
		posArray[index] = make_float4(pos, posData.w);
		velArray[index] = make_float4(vel, velData.w);
		velLeapFrogArray[index] = make_float4(velLeapFrog, velLeapFrogData.w);
}
