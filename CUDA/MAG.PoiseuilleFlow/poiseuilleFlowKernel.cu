#include "hip/hip_runtime.h"
#include <stdio.h>
#include <math.h>
#include "cutil_math.h"
#include "hip/hip_math_constants.h"
#include "poiseuilleFlowKernel.cuh"

#if USE_TEX
texture<float4, 1, hipReadModeElementType> oldPosTex;
texture<float4, 1, hipReadModeElementType> oldVelTex;
texture<float4, 1, hipReadModeElementType> oldMeasuresTex;

texture<uint, 1, hipReadModeElementType> gridParticleHashTex;
texture<uint, 1, hipReadModeElementType> cellStartTex;
texture<uint, 1, hipReadModeElementType> cellEndTex;
#endif
__constant__ PoiseuilleParams params;

__device__ int3 calcGridPos(float3 p){
	int3 gridPos;
	gridPos.x = floor((p.x - params.worldOrigin.x) * 0.5f / params.particleRadius);
	gridPos.y = floor((p.y - params.worldOrigin.y) * 0.5f / params.particleRadius);
	gridPos.z = floor((p.z - params.worldOrigin.z) * 0.5f / params.particleRadius);
	return gridPos;
}

__device__ uint calcGridHash(int3 gridPos){
	gridPos.x = gridPos.x & (params.gridSize.x-1);  
	gridPos.y = gridPos.y & (params.gridSize.y-1);
	gridPos.z = gridPos.z & (params.gridSize.z-1);        
	return __umul24(__umul24(gridPos.z, params.gridSize.y), params.gridSize.x) + __umul24(gridPos.y, params.gridSize.x) + gridPos.x;
}

__global__ void calculatePoiseuilleHashD(
	uint*   gridParticleHash,  // output
	uint*   gridParticleIndex, // output
	float4* pos,               // input
	uint    numParticles){
		uint index = __umul24(blockIdx.x, blockDim.x) + threadIdx.x;
		if (index >= numParticles) return;
	    
		volatile float4 p = pos[index];

		int3 gridPos = calcGridPos(make_float3(p.x, p.y, p.z));
		uint hash = calcGridHash(gridPos);

		gridParticleHash[index] = hash;
		gridParticleIndex[index] = index;
}

__global__ void reorderPoiseuilleDataD(
	uint*   cellStart,        // output
	uint*   cellEnd,          // output
	float4* sortedPos,        // output
	float4* sortedVel,        // output
	uint *  gridParticleHash, // input
	uint *  gridParticleIndex,// input
	float4* oldPos,           // input
	float4* oldVel,           // input
	uint    numParticles){
		extern __shared__ uint sharedHash[];    // blockSize + 1 elements
		uint index = __umul24(blockIdx.x,blockDim.x) + threadIdx.x;
		
		uint hash;
		if (index < numParticles) {
			hash = gridParticleHash[index];

			sharedHash[threadIdx.x+1] = hash;

			if (index > 0 && threadIdx.x == 0)
			{
				sharedHash[0] = gridParticleHash[index-1];
			}
		}

		__syncthreads();
		
		if (index < numParticles) {
			if (index == 0 || hash != sharedHash[threadIdx.x])
			{
				cellStart[hash] = index;
				if (index > 0)
					cellEnd[sharedHash[threadIdx.x]] = index;
			}

			if (index == numParticles - 1)
			{
				cellEnd[hash] = index + 1;
			}

			uint sortedIndex = gridParticleIndex[index];
			float4 pos = FETCH(oldPos, sortedIndex);       
			float4 vel = FETCH(oldVel, sortedIndex);       

			sortedPos[index] = pos;
			sortedVel[index] = vel;
		}
}

__device__ float sumParticlesInDomain(
	int3    gridPos,
	uint    index,
	float4  pos,
	float4* oldPos, 
	float4  vel,
	float4* oldVel,
	float4* measures,
	uint*   cellStart,
	uint*   cellEnd){
		uint gridHash = calcGridHash(gridPos);
		uint startIndex = FETCH(cellStart, gridHash);

		float sum = 0.0f;
		if (startIndex != 0xffffffff) {        // cell is not empty
			uint endIndex = FETCH(cellEnd, gridHash);
			for(uint j=startIndex; j<endIndex; j++) {				  
					float4 pos2 = FETCH(oldPos, j);
					float4 vel2 = FETCH(oldVel, j);
					float density2 = measures[j].x;
					float temp = 0.0f;

					float worldXSize= params.gridSize.x * 2.0f * params.particleRadius;				
					float3 relPos = make_float3(pos - pos2);
					if(gridPos.x < 0)
						relPos = make_float3(pos.x - (pos2.x - worldXSize),pos.y - pos2.y,pos.z - pos2.z); 
					else
						if(gridPos.x > params.gridSize.x - 1)
							relPos = make_float3(pos.x - (pos2.x + worldXSize),pos.y - pos2.y,pos.z - pos2.z);
						
					float dist = length(relPos);
					float q = dist / params.smoothingRadius;					
					
					float coeff = 7.0f / 478.0f / HIP_PI_F / powf(params.smoothingRadius, 2);
					if(q < 1){
						sum += coeff * (powf(3 - q, 5) - 6 * powf(2 - q, 5) + 15 * powf(1 - q, 5));	
						//sum +=1;
						continue;
					}
					if(q < 2){
						sum += coeff * (powf(3 - q, 5) - 6 * powf(2 - q, 5));
						//sum +=1;
						continue;
					} 
					if(q < 3){
						sum += coeff * powf(3 - q, 5);						
						//sum +=1;
						continue;
					}										               				
			}
		}
		return sum;
}

__global__ void calculatePoiseuilleDensityD(			
	float4* measures, //output
	float4* oldPos,	  //input 
	float4* oldVel,
	uint* gridParticleIndex,
	uint* cellStart,
	uint* cellEnd,
	uint numParticles){
		uint index = __mul24(blockIdx.x,blockDim.x) + threadIdx.x;
		if (index >= numParticles) return;    

		float4 pos = FETCH(oldPos, index);
		float4 vel = FETCH(oldVel, index);
		if(pos.w == 1.0f){				
				measures[index].x = params.restDensity;	
				measures[index].y = powf(params.soundspeed, 2) * params.restDensity;
				return;
		}
		int3 gridPos = calcGridPos(make_float3(pos));

		float sum = 0.0f;		
		for(int z=-params.cellcount; z<=params.cellcount; z++) {
			for(int y=-params.cellcount; y<=params.cellcount; y++) {
				for(int x=-params.cellcount; x<=params.cellcount; x++) {
					int3 neighbourPos = gridPos + make_int3(x, y, z);
					sum += sumParticlesInDomain(
							neighbourPos,
							index,
							pos,
							oldPos,
							vel,
							oldVel,
							measures,
							cellStart,
							cellEnd);
				}
			}
		}			
		float dens = sum * params.particleMass;
		measures[index].x = dens;	
		measures[index].y = powf(params.soundspeed, 2) * dens; 			
}

__device__ float4 getVelocityDiff(
	float4 iVelocity, 
	float4 iPosition, 
	float4 jVelocity,
	float4 jPosition)
{
	float bottomBoundary = params.worldOrigin.y + params.boundaryOffset * 2.0f * params.particleRadius;	
	float topBoundary = bottomBoundary + params.fluidParticlesSize.y * 2.0f * params.particleRadius;		
	if((jPosition.w == 1.0f) && (jPosition.y > topBoundary))
	{
		float distanceA = topBoundary - iPosition.y;
		float distanceB = jPosition.y - topBoundary;
		float beta = fmin(1.5f, 1 + distanceB / distanceA);
		return beta * iVelocity;
	}
	
	if((jPosition.w == 1.0f) && (jPosition.y < bottomBoundary))
	{
		float distanceA = iPosition.y - bottomBoundary;
		float distanceB = bottomBoundary - jPosition.y;
		float beta = fmin(1.5f, 1 + distanceB / distanceA);
		return beta * iVelocity;
	}
	return iVelocity - jVelocity;
}

__device__ float3 sumNavierStokesForces(
	int3    gridPos,
	uint    index,
	float4  pos,
	float4* oldPos, 
	float4  vel,
	float4* oldVel,
	float density,
	float pressure,				   
	float4* oldMeasures,
	uint*   cellStart,
	uint*   cellEnd){
		uint gridHash = calcGridHash(gridPos);
		uint startIndex = FETCH(cellStart, gridHash);
	    
		float3 tmpForce = make_float3(0.0f);
		float texp = 0.0f;
		float pexp = 0.0f;
		if (startIndex != 0xffffffff) {               
			uint endIndex = FETCH(cellEnd, gridHash);
			for(uint j=startIndex; j<endIndex; j++) {
				if (j != index) {             
					float4 pos2 = FETCH(oldPos, j);
					float4 vel2 = FETCH(oldVel, j);				
					float4 measure = FETCH(oldMeasures, j);
					float density2 = measure.x;
					float pressure2 = measure.y;				
					float tempExpr = 0.0f;

					float worldXSize= params.gridSize.x * 2.0f * params.particleRadius;				
					float3 relPos = make_float3(pos - pos2);
					if(gridPos.x < 0)
						relPos = make_float3(pos) - make_float3(pos2.x - worldXSize, pos2.y, pos2.z); 
					else
						if(gridPos.x > params.gridSize.x - 1)
							relPos = make_float3(pos) - make_float3(pos2.x + worldXSize, pos2.y, pos2.z); 
										
					float dist = length(relPos);
					float q = dist / params.smoothingRadius;				

					float coeff = -7.0f / 478.0f / HIP_PI_F / powf(params.smoothingRadius, 3);
					float temp = 0.0f;
					float4 Vab = getVelocityDiff(vel, pos, vel2, pos2);
					if(q < 1){
						temp = coeff * (5 * powf(3 - q, 4) - 30 * powf(2 - q, 4) + 75 * powf(1 - q, 4));	
						tmpForce += -1.0f * params.particleMass *
							(pressure / powf(density,2) + pressure2 / powf(density2,2)) * 
							normalize(relPos) * temp +
							params.particleMass * (params.mu + params.mu) * 
							make_float3(Vab) / (density * density2) * 1.0f / dist * temp;
						continue;
					}
					if(q < 2){
						temp = coeff * (5 * powf(3 - q, 4) - 30 * powf(2 - q, 4));
						tmpForce += -1.0f * params.particleMass *
							(pressure / powf(density,2) + pressure2 / powf(density2,2)) * 
							normalize(relPos) * temp +
							params.particleMass * (params.mu + params.mu) * 
							make_float3(Vab) / (density * density2) * 1.0f / dist * temp;
						continue;
					} 
					if(q < 3){
						temp = coeff * (5 * powf(3 - q, 4));
						tmpForce += -1.0f * params.particleMass *
							(pressure / powf(density,2) + pressure2 / powf(density2,2)) * 
							normalize(relPos) * temp +
							params.particleMass * (params.mu + params.mu) * 
							make_float3(Vab) / (density * density2) * 1.0f / dist * temp;
						continue;
					}	
				}
			}
		}
		return tmpForce;				
}

__global__ void calculatePoiseuilleAccelerationD(
	float4* acceleration,			
	float4* oldMeasures,
	float4* oldPos,			
	float4* oldVel,
	uint* gridParticleIndex,
	uint* cellStart,
	uint* cellEnd,
	uint numParticles){
		uint index = __mul24(blockIdx.x,blockDim.x) + threadIdx.x;
		if (index >= numParticles) return;    

		float4 pos = FETCH(oldPos, index);
		float4 vel = FETCH(oldVel, index);
		float4 measure = FETCH(oldMeasures,index);
		float density = measure.x;
		float pressure = measure.y;

		int3 gridPos = calcGridPos(make_float3(pos));

		float3 force = make_float3(0.0f);		
		for(int z=-params.cellcount; z<=params.cellcount; z++) {
			for(int y=-params.cellcount; y<=params.cellcount; y++) {
				for(int x=-params.cellcount; x<=params.cellcount; x++) {
					int3 neighbourPos = gridPos + make_int3(x, y, z);
					force += sumNavierStokesForces(neighbourPos, 
						index, 
						pos, 
						oldPos,
						vel,
						oldVel,
						density,
						pressure,					
						oldMeasures,
						cellStart, 
						cellEnd);
				}
			}
		}
		uint originalIndex = gridParticleIndex[index];					
		float3 acc = force;			
		acceleration[originalIndex] = make_float4(acc, 0.0f);
}

__global__ void integratePoiseuilleSystemD(
	float4* posArray,		 // input, output
	float4* velArray,		 // input, output  
	float4* velLeapFrogArray, // output
	float4* acceleration,	 // input
	uint numParticles){
		uint index = __umul24(blockIdx.x,blockDim.x) + threadIdx.x;
		if (index >= numParticles) return;          

		volatile float4 posData = posArray[index]; 
		if(posData.w == 1.0f) return;//skip boundary particle

		volatile float4 velData = velArray[index];
		volatile float4 accData = acceleration[index];
		volatile float4 velLeapFrogData = velLeapFrogArray[index];

		float3 pos = make_float3(posData.x, posData.y, posData.z);
		float3 vel = make_float3(velData.x, velData.y, velData.z);
		float3 acc = make_float3(accData.x, accData.y, accData.z);
		
		float3 nextVel = vel + (params.gravity + acc) * params.deltaTime;						

		float3 velLeapFrog = vel + nextVel;
		velLeapFrog *= 0.5f;

		vel = nextVel;   	
		pos += vel * params.deltaTime;   

		float halfWorldXSize = params.gridSize.x * params.particleRadius;			

		if(pos.x > halfWorldXSize){
			pos.x -= 2 * halfWorldXSize;
		}
		  
		posArray[index] = make_float4(pos, posData.w);
		velArray[index] = make_float4(vel, velData.w);
		velLeapFrogArray[index] = make_float4(velLeapFrog, velLeapFrogData.w);
}
