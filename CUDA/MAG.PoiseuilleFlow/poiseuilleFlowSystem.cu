#include "hip/hip_runtime.h"
#include <cutil_inline.h>
#include <cstdlib>
#include <cstdio>
#include <string.h>
#include <GL/glut.h>
#include <cuda_gl_interop.h>
#include "poiseuilleFlowKernel.cu"
#include "magUtil.cuh"
extern "C"
{	
	void setParameters(PoiseuilleParams *hostParams){
		cutilSafeCall( hipMemcpyToSymbol(HIP_SYMBOL(params), hostParams, sizeof(PoiseuilleParams)) );
	}		

	void integratePoiseuilleSystem(
		float *pos,
		float *vel,  
		float* velLeapFrog,
		float *acc,
		uint numParticles){
			uint numThreads, numBlocks;
			computeGridSize(numParticles, 256, numBlocks, numThreads);

			integratePoiseuilleSystemD<<< numBlocks, numThreads >>>(
				(float4*)pos,
				(float4*)vel,
				(float4*)velLeapFrog,
				(float4*)acc,
				numParticles);
		    
			cutilCheckMsg("integrate kernel execution failed");
	}

	void calculatePoiseuilleHash(
		uint* gridParticleHash,
		uint* gridParticleIndex,
		float* pos, 
		int numParticles){
			uint numThreads, numBlocks;
			computeGridSize(numParticles, 256, numBlocks, numThreads);

			calculatePoiseuilleHashD<<< numBlocks, numThreads >>>(
				gridParticleHash,
				gridParticleIndex,
				(float4 *) pos,
				numParticles);
		    
			cutilCheckMsg("Kernel execution failed: calculatePoiseuilleHashD");
	}

	void reorderPoiseuilleData(
		uint*  cellStart,
		uint*  cellEnd,
		float* sortedPos,
		float* sortedVel,
		uint*  gridParticleHash,
		uint*  gridParticleIndex,
		float* oldPos,
		float* oldVel,
		uint   numParticles,
		uint   numCells){
			uint numThreads, numBlocks;
			computeGridSize(numParticles, 256, numBlocks, numThreads);

			cutilSafeCall(hipMemset(cellStart, 0xffffffff, numCells*sizeof(uint)));

			#if USE_TEX
				cutilSafeCall(hipBindTexture(0, oldPosTex, oldPos, numParticles*sizeof(float4)));
				cutilSafeCall(hipBindTexture(0, oldVelTex, oldVel, numParticles*sizeof(float4)));
			#endif

				uint smemSize = sizeof(uint)*(numThreads+1);
				reorderPoiseuilleDataD<<< numBlocks, numThreads, smemSize>>>(
					cellStart,
					cellEnd,
					(float4 *) sortedPos,
					(float4 *) sortedVel,
					gridParticleHash,
					gridParticleIndex,
					(float4 *) oldPos,
					(float4 *) oldVel,
					numParticles);
				cutilCheckMsg("Kernel execution failed: reorderPoiseuilleDataD");

			#if USE_TEX
				cutilSafeCall(hipUnbindTexture(oldPosTex));
				cutilSafeCall(hipUnbindTexture(oldVelTex));
			#endif
	}

	void calculatePoiseuilleDensity(			
		float* measures,
		float* sortedPos,	
		float* sortedVel,
		uint* gridParticleIndex,
		uint* cellStart,
		uint* cellEnd,
		uint numParticles,
		uint numGridCells){
			#if USE_TEX
			cutilSafeCall(hipBindTexture(0, oldPosTex, sortedPos, numParticles*sizeof(float4)));
			cutilSafeCall(hipBindTexture(0, oldVelTex, sortedVel, numParticles*sizeof(float4)));
			cutilSafeCall(hipBindTexture(0, cellStartTex, cellStart, numGridCells*sizeof(uint)));
			cutilSafeCall(hipBindTexture(0, cellEndTex, cellEnd, numGridCells*sizeof(uint)));    
			#endif

			uint numThreads, numBlocks;
			computeGridSize(numParticles, 64, numBlocks, numThreads);

			calculatePoiseuilleDensityD<<< numBlocks, numThreads >>>(										  
				(float4*)measures,
				(float4*)sortedPos,  
				(float4*)sortedVel,
				gridParticleIndex,
				cellStart,
				cellEnd,
				numParticles);

			cutilCheckMsg("Kernel execution failed");

			#if USE_TEX
			cutilSafeCall(hipUnbindTexture(oldPosTex));
			cutilSafeCall(hipUnbindTexture(oldVelTex));
			cutilSafeCall(hipUnbindTexture(cellStartTex));
			cutilSafeCall(hipUnbindTexture(cellEndTex));
			#endif
	}

	void calculatePoiseuilleAcceleration(
		float* acceleration,
		float* sortedMeasures,			
		float* sortedPos,			
		float* sortedVel,
		uint* gridParticleIndex,
		uint* cellStart,
		uint* cellEnd,
		uint numParticles,
		uint numGridCells){
			#if USE_TEX
			cutilSafeCall(hipBindTexture(0, oldPosTex, sortedPos, numParticles*sizeof(float4)));
			cutilSafeCall(hipBindTexture(0, oldVelTex, sortedVel, numParticles*sizeof(float4)));
			cutilSafeCall(hipBindTexture(0, oldMeasuresTex, sortedMeasures, numParticles*sizeof(float4)));
			cutilSafeCall(hipBindTexture(0, cellStartTex, cellStart, numGridCells*sizeof(uint)));
			cutilSafeCall(hipBindTexture(0, cellEndTex, cellEnd, numGridCells*sizeof(uint)));    
			#endif

			uint numThreads, numBlocks;
			computeGridSize(numParticles, 64, numBlocks, numThreads);

			calculatePoiseuilleAccelerationD<<< numBlocks, numThreads >>>(
				(float4*)acceleration,
				(float4*)sortedMeasures,										  
				(float4*)sortedPos,                                          
				(float4*)sortedVel, 
				gridParticleIndex,
				cellStart,
				cellEnd,
				numParticles);

			cutilCheckMsg("Kernel execution failed");

			#if USE_TEX
			cutilSafeCall(hipUnbindTexture(oldPosTex));
			cutilSafeCall(hipUnbindTexture(oldVelTex));
			cutilSafeCall(hipUnbindTexture(oldMeasuresTex));
			cutilSafeCall(hipUnbindTexture(cellStartTex));
			cutilSafeCall(hipUnbindTexture(cellEndTex));
			#endif
	}
}// extern "C"

