#include "hip/hip_runtime.h"
#include "cutil_math.h"
#include "poiseulleFlowUtil.cu"

__device__ float4 getVelocityDiff(
	float4 iVelocity, 
	float4 iPosition, 
	float4 jVelocity,
	float4 jPosition,
	float elapsedTime)
{		
	/*float bottomBoundary = params.worldOrigin.y + params.boundaryOffset * 2.0f * params.particleRadius + params.amplitude;	
	float topBoundary = bottomBoundary + params.fluidParticlesSize.y * 2.0f * params.particleRadius ;		
	if(jPosition.w < 0.0f)
	{
		float distanceA = topBoundary - iPosition.y;
		float distanceB = jPosition.y - topBoundary;
		float beta = fmin(1000.0f, 1 + distanceB / distanceA);
		return beta * iVelocity;
	}
	
	if(jPosition.w > 0.0f)
	{
		float distanceA = iPosition.y - bottomBoundary;
		float distanceB = bottomBoundary - jPosition.y;
		float beta = fmin(1000.0f, 1 + distanceB / distanceA);
		return beta * iVelocity;
	}*/
	return iVelocity - jVelocity;	
}

__device__ float3 sumViscosity(
	int3    gridPos,
	uint    index,
	float4  pos,
	float4* oldPos, 
	float4  vel,
	float4* oldVel,
	float density,
	float4* oldMeasures,
	uint*   cellStart,
	uint*   cellEnd,
	float elapsedTime){
		uint gridHash = calcGridHash(gridPos);
		int3 shift = make_int3(EvaluateShift(gridPos.x, params.gridSize.x),
			EvaluateShift(gridPos.y, params.gridSize.y),
			EvaluateShift(gridPos.z, params.gridSize.z));							

		uint startIndex = FETCH(cellStart, gridHash);	    
		float3 force = make_float3(0.0f);
		if (startIndex != 0xffffffff) {               
			uint endIndex = FETCH(cellEnd, gridHash);
			for(uint j=startIndex; j<endIndex; j++) {
				if (j != index) {             
					float4 pos2 = FETCH(oldPos, j);
					float4 vel2 = FETCH(oldVel, j);
					float4 measure = FETCH(oldMeasures, j);
					float density2 = measure.x;

					float3 relPos = make_float3(pos.x - (pos2.x + shift.x * params.worldSize.x),
						pos.y - (pos2.y + shift.y * params.worldSize.y),
						pos.z - (pos2.z + shift.z * params.worldSize.z));
					  										
					float dist = length(relPos);
					float q = dist / params.smoothingRadius;									

					float coeff = 7.0f / (2 * HIP_PI_F * powf(params.smoothingRadius, 3));
					float4 Vab = getVelocityDiff(vel, pos, vel2, pos2, elapsedTime);
					if(q < 2){
						float temp = coeff * (-powf(1 - 0.5f * q,3) * (2 * q + 1) + powf(1 - 0.5f * q, 4));
						force += params.particleMass * (params.mu + params.mu) * 
							make_float3(Vab) / (density * density2 *  dist) * temp;							
					}
				}
			}
		}
		return force;				
}

__global__ void computeViscousForceD(
	float4* viscousForce,	
	float4* oldMeasures,
	float4* oldPos,			
	float4* oldVel,
	uint* gridParticleIndex,
	uint* cellStart,
	uint* cellEnd,
	uint numParticles,
	float elapsedTime){
		uint index = __mul24(blockIdx.x,blockDim.x) + threadIdx.x;
		if (index >= numParticles) return;    

		float4 pos = FETCH(oldPos, index);
		float4 vel = FETCH(oldVel, index);
		float4 measure = FETCH(oldMeasures,index);
		float density = measure.x;

		int3 gridPos = calcGridPos(make_float3(pos));

		float3 force = make_float3(0.0f);		
		for(int z=-params.cellcount; z<=params.cellcount; z++) {
			for(int y=-params.cellcount; y<=params.cellcount; y++) {
				for(int x=-params.cellcount; x<=params.cellcount; x++) {
					int3 neighbourPos = gridPos + make_int3(x, y, z);
					force += sumViscosity(
						neighbourPos, 
						index, 
						pos, 
						oldPos,
						vel,
						oldVel,
						density,
						oldMeasures,
						cellStart, 
						cellEnd,
						elapsedTime);
				}
			}
		}
		uint originalIndex = gridParticleIndex[index];							
		viscousForce[originalIndex] = make_float4(force, 0.0f);
}