#include "hip/hip_runtime.h"
#include <cutil_inline.h>
#include <GL/freeglut.h>
#include <cuda_gl_interop.h>
typedef unsigned int uint;

extern "C"
{
	uint iDivUp(uint a, uint b){
		return (a % b != 0) ? (a / b + 1) : (a / b);
	}

	void computeGridSize(uint n, uint blockSize, uint &numBlocks, uint &numThreads){
		numThreads = min(blockSize, n);
		numBlocks = iDivUp(n, numThreads);
	}

	void cudaInit(int argc, char **argv)
	{   		
		if( cutCheckCmdLineFlag(argc, (const char**)argv, "device") ) {
			cutilDeviceInit(argc, argv);
		} else {
			hipSetDevice( cutGetMaxGflopsDeviceId() );
		}
	}

	void cudaGLInit(int argc, char **argv)
	{   
		if( cutCheckCmdLineFlag(argc, (const char**)argv, "device") ) {
			cutilDeviceInit(argc, argv);
		} else {
			cudaGLSetGLDevice( cutGetMaxGflopsDeviceId() );
		}
	}

	void registerGLBufferObject(uint vbo, struct hipGraphicsResource **cuda_vbo_resource)
	{
		cutilSafeCall(hipGraphicsGLRegisterBuffer(cuda_vbo_resource, vbo, 
							   cudaGraphicsMapFlagsNone));
	}

	void unregisterGLBufferObject(struct hipGraphicsResource *cuda_vbo_resource)
	{
		cutilSafeCall(hipGraphicsUnregisterResource(cuda_vbo_resource));	
	}

	void *mapGLBufferObject(struct hipGraphicsResource **cuda_vbo_resource)
	{
		void *ptr;
		cutilSafeCall(hipGraphicsMapResources(1, cuda_vbo_resource, 0));
		size_t num_bytes; 
		cutilSafeCall(hipGraphicsResourceGetMappedPointer((void **)&ptr, &num_bytes,  
								   *cuda_vbo_resource));
		return ptr;
	}

	void unmapGLBufferObject(struct hipGraphicsResource *cuda_vbo_resource)
	{
	   cutilSafeCall(hipGraphicsUnmapResources(1, &cuda_vbo_resource, 0));
	}

	void allocateArray(void **devPtr, size_t size)
	{
		cutilSafeCall(hipMalloc(devPtr, size));
	}

	void freeArray(void *devPtr)
	{
		cutilSafeCall(hipFree(devPtr));
	}

	void copyArrayFromDevice(void* host, const void* device, 
			 struct hipGraphicsResource **cuda_vbo_resource, int size)
	{   
		if (cuda_vbo_resource)
		device = mapGLBufferObject(cuda_vbo_resource);

		cutilSafeCall(hipMemcpy(host, device, size, hipMemcpyDeviceToHost));
	    
		if (cuda_vbo_resource)
		unmapGLBufferObject(*cuda_vbo_resource);
	}

	void copyArrayToDevice(void* device, const void* host, int offset, int size)
	{
		cutilSafeCall(hipMemcpy((char *) device + offset, host, size, hipMemcpyHostToDevice));
	}

	
}