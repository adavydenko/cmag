#include "hip/hip_runtime.h"
#include <cutil_inline.h>
#include <cstdlib>
#include <cstdio>
#include <string.h>
#include <GL/freeglut.h>
#include <cuda_gl_interop.h>
#include "thrust/device_ptr.h"
#include "thrust/for_each.h"
#include "thrust/iterator/zip_iterator.h"
#include "thrust/sort.h"
#include "fluid_kernel.cu"
extern "C"
{		
	void setParameters(SimParams *hostParams){
		cutilSafeCall( hipMemcpyToSymbol(HIP_SYMBOL(params), hostParams, sizeof(SimParams)) );
	}

	uint iDivUp(uint a, uint b){
		return (a % b != 0) ? (a / b + 1) : (a / b);
	}

	void computeGridSize(uint n, uint blockSize, uint &numBlocks, uint &numThreads){
		numThreads = min(blockSize, n);
		numBlocks = iDivUp(n, numThreads);
	}

	void cudaGLInit(int argc, char **argv)
	{   
		if( cutCheckCmdLineFlag(argc, (const char**)argv, "device") ) {
			cutilDeviceInit(argc, argv);
		} else {
			cudaGLSetGLDevice( cutGetMaxGflopsDeviceId() );
		}
	}

	void registerGLBufferObject(uint vbo, struct hipGraphicsResource **cuda_vbo_resource)
	{
		cutilSafeCall(hipGraphicsGLRegisterBuffer(cuda_vbo_resource, vbo, 
							   cudaGraphicsMapFlagsNone));
	}

	void unregisterGLBufferObject(struct hipGraphicsResource *cuda_vbo_resource)
	{
		cutilSafeCall(hipGraphicsUnregisterResource(cuda_vbo_resource));	
	}

	void *mapGLBufferObject(struct hipGraphicsResource **cuda_vbo_resource)
	{
		void *ptr;
		cutilSafeCall(hipGraphicsMapResources(1, cuda_vbo_resource, 0));
		size_t num_bytes; 
		cutilSafeCall(hipGraphicsResourceGetMappedPointer((void **)&ptr, &num_bytes,  
								   *cuda_vbo_resource));
		return ptr;
	}

	void unmapGLBufferObject(struct hipGraphicsResource *cuda_vbo_resource)
	{
	   cutilSafeCall(hipGraphicsUnmapResources(1, &cuda_vbo_resource, 0));
	}

	void allocateArray(void **devPtr, size_t size)
	{
		cutilSafeCall(hipMalloc(devPtr, size));
	}

	void freeArray(void *devPtr)
	{
		cutilSafeCall(hipFree(devPtr));
	}

	void copyArrayToDevice(void* device, const void* host, int offset, int size)
	{
		cutilSafeCall(hipMemcpy((char *) device + offset, host, size, hipMemcpyHostToDevice));
	}

	void ExtChangeRightBoundary(
		float * position,
		uint numParticles){
			uint numThreads, numBlocks;
			computeGridSize(numParticles, 256, numBlocks, numThreads);

			shiftRightBoundaryD<<< numBlocks, numThreads >>>(
				(float4*)position,
				numParticles);
		    
			cutilCheckMsg("removeRightBoundary kernel execution failed");
	}

	void sortParticles(uint *dHash, uint *dIndex, uint numParticles)
	{
		thrust::sort_by_key(thrust::device_ptr<uint>(dHash),
							thrust::device_ptr<uint>(dHash + numParticles),
							thrust::device_ptr<uint>(dIndex));
	}

	void ExtRemoveRightBoundary(
		float * position,
		uint numParticles){
			uint numThreads, numBlocks;
			computeGridSize(numParticles, 256, numBlocks, numThreads);

			removeRightBoundaryD<<< numBlocks, numThreads >>>(
				(float4*)position,
				numParticles);
		    
			cutilCheckMsg("removeRightBoundary kernel execution failed");
	}

	void integrateSystem(
		float *pos,
		float *vel,  
		float* velLeapFrog,
		float *acc,
		uint numParticles){
			uint numThreads, numBlocks;
			computeGridSize(numParticles, 256, numBlocks, numThreads);

			integrate<<< numBlocks, numThreads >>>(
				(float4*)pos,
				(float4*)vel,
				(float4*)velLeapFrog,
				(float4*)acc,
				numParticles);
		    
			cutilCheckMsg("integrate kernel execution failed");
	}

	void calcHash(
		uint* gridParticleHash,
		uint* gridParticleIndex,
		float* pos, 
		int numParticles){
			uint numThreads, numBlocks;
			computeGridSize(numParticles, 256, numBlocks, numThreads);

			calcHashD<<< numBlocks, numThreads >>>(
				gridParticleHash,
				gridParticleIndex,
				(float4 *) pos,
				numParticles);
		    
			cutilCheckMsg("Kernel execution failed");
	}

	void reorderDataAndFindCellStart(
		uint*  cellStart,
		uint*  cellEnd,
		float* sortedPos,
		float* sortedVel,
		uint*  gridParticleHash,
		uint*  gridParticleIndex,
		float* oldPos,
		float* oldVel,
		uint   numParticles,
		uint   numCells){
			uint numThreads, numBlocks;
			computeGridSize(numParticles, 256, numBlocks, numThreads);

			cutilSafeCall(hipMemset(cellStart, 0xffffffff, numCells*sizeof(uint)));

			#if USE_TEX
				cutilSafeCall(hipBindTexture(0, oldPosTex, oldPos, numParticles*sizeof(float4)));
				cutilSafeCall(hipBindTexture(0, oldVelTex, oldVel, numParticles*sizeof(float4)));
			#endif

				uint smemSize = sizeof(uint)*(numThreads+1);
				reorderDataAndFindCellStartD<<< numBlocks, numThreads, smemSize>>>(
					cellStart,
					cellEnd,
					(float4 *) sortedPos,
					(float4 *) sortedVel,
					gridParticleHash,
					gridParticleIndex,
					(float4 *) oldPos,
					(float4 *) oldVel,
					numParticles);
				cutilCheckMsg("Kernel execution failed: reorderDataAndFindCellStartD");

			#if USE_TEX
				cutilSafeCall(hipUnbindTexture(oldPosTex));
				cutilSafeCall(hipUnbindTexture(oldVelTex));
			#endif
	}	

	void calculateDamBreakDensity(			
		float* sortedMeasuresOutput,
		float* sortedMeasures,
		float* sortedPos,			
		float* sortedVel,		
		uint* gridParticleIndex,
		uint* cellStart,
		uint* cellEnd,
		uint numParticles,
		uint numGridCells){
			#if USE_TEX
			cutilSafeCall(hipBindTexture(0, oldPosTex, sortedPos, numParticles*sizeof(float4)));
			cutilSafeCall(hipBindTexture(0, oldMeasuresTex, sortedMeasures, numParticles*sizeof(float4)));
			cutilSafeCall(hipBindTexture(0, oldVelTex, sortedVel, numParticles*sizeof(float4)));
			cutilSafeCall(hipBindTexture(0, cellStartTex, cellStart, numGridCells*sizeof(uint)));
			cutilSafeCall(hipBindTexture(0, cellEndTex, cellEnd, numGridCells*sizeof(uint)));    
			#endif

			uint numThreads, numBlocks;
			computeGridSize(numParticles, 64, numBlocks, numThreads);

			calculateDamBreakDensityD<<< numBlocks, numThreads >>>(										  
				(float4*)sortedMeasuresOutput,
				(float4*)sortedMeasures,
				(float4*)sortedPos,                                          
				(float4*)sortedVel, 
				gridParticleIndex,
				cellStart,
				cellEnd,
				numParticles);

			cutilCheckMsg("Kernel execution failed");

			#if USE_TEX
			cutilSafeCall(hipUnbindTexture(oldPosTex));
			cutilSafeCall(hipUnbindTexture(oldMeasuresTex));
			cutilSafeCall(hipUnbindTexture(oldVelTex));
			cutilSafeCall(hipUnbindTexture(cellStartTex));
			cutilSafeCall(hipUnbindTexture(cellEndTex));			
			#endif
	}

	void calcAndApplyAcceleration(
		float* acceleration,
		float* sortedMeasures,			
		float* sortedPos,			
		float* sortedVel,
		uint* gridParticleIndex,
		uint* cellStart,
		uint* cellEnd,
		uint numParticles,
		uint numGridCells){
			#if USE_TEX
			cutilSafeCall(hipBindTexture(0, oldPosTex, sortedPos, numParticles*sizeof(float4)));
			cutilSafeCall(hipBindTexture(0, oldVelTex, sortedVel, numParticles*sizeof(float4)));
			cutilSafeCall(hipBindTexture(0, oldMeasuresTex, sortedMeasures, numParticles*sizeof(float4)));
			cutilSafeCall(hipBindTexture(0, cellStartTex, cellStart, numGridCells*sizeof(uint)));
			cutilSafeCall(hipBindTexture(0, cellEndTex, cellEnd, numGridCells*sizeof(uint)));    
			#endif

			uint numThreads, numBlocks;
			computeGridSize(numParticles, 64, numBlocks, numThreads);

			calcAndApplyAccelerationD<<< numBlocks, numThreads >>>(
				(float4*)acceleration,
				(float4*)sortedMeasures,										  
				(float4*)sortedPos,                                          
				(float4*)sortedVel, 
				gridParticleIndex,
				cellStart,
				cellEnd,
				numParticles);

			cutilCheckMsg("Kernel execution failed");

			#if USE_TEX
			cutilSafeCall(hipUnbindTexture(oldPosTex));
			cutilSafeCall(hipUnbindTexture(oldVelTex));
			cutilSafeCall(hipUnbindTexture(oldMeasuresTex));
			cutilSafeCall(hipUnbindTexture(cellStartTex));
			cutilSafeCall(hipUnbindTexture(cellEndTex));
			#endif
	}
}// extern "C"

