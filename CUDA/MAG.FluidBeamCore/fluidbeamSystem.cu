#include "hip/hip_runtime.h"
#include <cutil_inline.h>
#include <cstdlib>
#include <cstdio>
#include <string.h>
#include <GL/glut.h>
#include <cuda_gl_interop.h>
#include "fluidbeam_kernel.cu"

extern "C"
{

void cudaInit(int argc, char **argv)
{   
    if( cutCheckCmdLineFlag(argc, (const char**)argv, "device") ) {
        cutilDeviceInit(argc, argv);
    } else {
        hipSetDevice( cutGetMaxGflopsDeviceId() );
    }
}

void cudaGLInit(int argc, char **argv)
{   
    if( cutCheckCmdLineFlag(argc, (const char**)argv, "device") ) {
        cutilDeviceInit(argc, argv);
    } else {
        cudaGLSetGLDevice( cutGetMaxGflopsDeviceId() );
    }
}

void allocateArray(void **devPtr, size_t size)
{
    cutilSafeCall(hipMalloc(devPtr, size));
}

void freeArray(void *devPtr)
{
    cutilSafeCall(hipFree(devPtr));
}

void threadSync()
{
    cutilSafeCall(hipDeviceSynchronize());
}

void copyArrayToDevice(void* device, const void* host, int offset, int size)
{
    cutilSafeCall(hipMemcpy((char *) device + offset, host, size, hipMemcpyHostToDevice));
}

void registerGLBufferObject(uint vbo, struct hipGraphicsResource **cuda_vbo_resource)
{
    cutilSafeCall(hipGraphicsGLRegisterBuffer(cuda_vbo_resource, vbo, 
					       cudaGraphicsMapFlagsNone));
}

void unregisterGLBufferObject(struct hipGraphicsResource *cuda_vbo_resource)
{
    cutilSafeCall(hipGraphicsUnregisterResource(cuda_vbo_resource));	
}

void *mapGLBufferObject(struct hipGraphicsResource **cuda_vbo_resource)
{
    void *ptr;
    cutilSafeCall(hipGraphicsMapResources(1, cuda_vbo_resource, 0));
    size_t num_bytes; 
    cutilSafeCall(hipGraphicsResourceGetMappedPointer((void **)&ptr, &num_bytes,  
						       *cuda_vbo_resource));
    return ptr;
}

void unmapGLBufferObject(struct hipGraphicsResource *cuda_vbo_resource)
{
   cutilSafeCall(hipGraphicsUnmapResources(1, &cuda_vbo_resource, 0));
}

void copyArrayFromDevice(void* host, const void* device, 
			 struct hipGraphicsResource **cuda_vbo_resource, int size)
{   
    if (cuda_vbo_resource)
	device = mapGLBufferObject(cuda_vbo_resource);

    cutilSafeCall(hipMemcpy(host, device, size, hipMemcpyDeviceToHost));
    
    if (cuda_vbo_resource)
	unmapGLBufferObject(*cuda_vbo_resource);
}

void setParameters(SimParams *hostParams)
{
    cutilSafeCall( hipMemcpyToSymbol(HIP_SYMBOL(params), hostParams, sizeof(SimParams)) );
}

uint iDivUp(uint a, uint b){
    return (a % b != 0) ? (a / b + 1) : (a / b);
}

void computeGridSize(uint n, uint blockSize, uint &numBlocks, uint &numThreads)
{
    numThreads = min(blockSize, n);
    numBlocks = iDivUp(n, numThreads);
}

void integrateSystem(float *pos,
                     float *vel,  
					 float* velLeapFrog,
					 float *acc,
                     uint numParticles)
{
    uint numThreads, numBlocks;
    computeGridSize(numParticles, 256, numBlocks, numThreads);

    integrate<<< numBlocks, numThreads >>>((float4*)pos,
                                           (float4*)vel,
										   (float4*)velLeapFrog,
										   (float4*)acc,
                                           numParticles);
    
    cutilCheckMsg("integrate kernel execution failed");
}

void calcHash(uint*  gridParticleHash,
              uint*  gridParticleIndex,
              float* pos, 
              int    numParticles)
{
    uint numThreads, numBlocks;
    computeGridSize(numParticles, 256, numBlocks, numThreads);

    calcHashD<<< numBlocks, numThreads >>>(gridParticleHash,
                                           gridParticleIndex,
                                           (float4 *) pos,
                                           numParticles);
    
    cutilCheckMsg("Kernel execution failed");
}

void reorderDataAndFindCellStart(uint*  cellStart,
							     uint*  cellEnd,
							     float* sortedPos,
								 float* sortedReferencePos,
							     float* sortedVel,
                                 uint*  gridParticleHash,
                                 uint*  gridParticleIndex,
							     float* oldPos,
								 float* oldReferencePos,
							     float* oldVel,
							     uint   numParticles,
							     uint   numCells)
{
    uint numThreads, numBlocks;
    computeGridSize(numParticles, 256, numBlocks, numThreads);

	cutilSafeCall(hipMemset(cellStart, 0xffffffff, numCells*sizeof(uint)));

	#if USE_TEX
		cutilSafeCall(hipBindTexture(0, oldPosTex, oldPos, numParticles*sizeof(float4)));
		cutilSafeCall(hipBindTexture(0, oldReferencePosTex, oldReferencePos, numParticles*sizeof(float4)));
		cutilSafeCall(hipBindTexture(0, oldVelTex, oldVel, numParticles*sizeof(float4)));
	#endif

		uint smemSize = sizeof(uint)*(numThreads+1);
		reorderDataAndFindCellStartD<<< numBlocks, numThreads, smemSize>>>(
			cellStart,
			cellEnd,
			(float4*) sortedPos,
			(float4*) sortedReferencePos,
			(float4*) sortedVel,
			gridParticleHash,
			gridParticleIndex,
			(float4*) oldPos,
			(float4*) oldReferencePos,
			(float4*) oldVel,
			numParticles);
		cutilCheckMsg("Kernel execution failed: reorderDataAndFindCellStartD");

	#if USE_TEX
		cutilSafeCall(hipUnbindTexture(oldPosTex));
		cutilSafeCall(hipUnbindTexture(oldReferencePosTex));
		cutilSafeCall(hipUnbindTexture(oldVelTex));
	#endif
}

void calcDensityAndPressure(			
			float* measures,
			float* sortedPos,			
			float* sortedVelocities,
			uint* gridParticleIndex,
			uint* cellStart,
			uint* cellEnd,
			uint numParticles,
			uint numGridCells)
{

	#if USE_TEX
    cutilSafeCall(hipBindTexture(0, oldPosTex, sortedPos, numParticles*sizeof(float4)));
	cutilSafeCall(hipBindTexture(0, oldVelTex, sortedVelocities, numParticles*sizeof(float4)));
    cutilSafeCall(hipBindTexture(0, cellStartTex, cellStart, numGridCells*sizeof(uint)));
    cutilSafeCall(hipBindTexture(0, cellEndTex, cellEnd, numGridCells*sizeof(uint)));    
	#endif

	uint numThreads, numBlocks;
    computeGridSize(numParticles, 64, numBlocks, numThreads);

    CalculateDensityAndPressureD<<< numBlocks, numThreads >>>(										  
										  (float4*)measures,
                                          (float4*)sortedPos,                                          
										  (float4*)sortedVelocities,
                                          gridParticleIndex,
                                          cellStart,
                                          cellEnd,
                                          numParticles);

    cutilCheckMsg("Kernel execution failed");

	#if USE_TEX
    cutilSafeCall(hipUnbindTexture(oldPosTex));
	cutilSafeCall(hipUnbindTexture(oldVelTex));
    cutilSafeCall(hipUnbindTexture(cellStartTex));
    cutilSafeCall(hipUnbindTexture(cellEndTex));
	#endif
}

void calcDisplacementGradient(
				float* udisplacementGradient, 
				float* vdisplacementGradient, 
				float* wdisplacementGradient, 
				float* sortedPos,	
				float* sortedReferencePos,						
				uint* Index,
				uint* cellStart,
				uint* cellEnd,
				uint numParticles,
				uint numGridCells)
	{
		#if USE_TEX
		cutilSafeCall(hipBindTexture(0, oldPosTex, sortedPos, numParticles*sizeof(float4)));
		cutilSafeCall(hipBindTexture(0, oldReferencePosTex, sortedReferencePos, numParticles*sizeof(float4)));		
		cutilSafeCall(hipBindTexture(0, cellStartTex, cellStart, numGridCells*sizeof(uint)));
		cutilSafeCall(hipBindTexture(0, cellEndTex, cellEnd, numGridCells*sizeof(uint)));    
		#endif

		uint numThreads, numBlocks;
		computeGridSize(numParticles, 64, numBlocks, numThreads);

		calcDisplacementGradientD<<< numBlocks, numThreads >>>(
			(float4*)udisplacementGradient, 
			(float4*)vdisplacementGradient, 
			(float4*)wdisplacementGradient, 
			(float4*)sortedPos,	
			(float4*)sortedReferencePos,				
			Index, 
			cellStart,
			cellEnd,
			numParticles);

		cutilCheckMsg("Kernel execution failed");

		#if USE_TEX
		cutilSafeCall(hipUnbindTexture(oldPosTex));
		cutilSafeCall(hipUnbindTexture(oldReferencePosTex));		
		cutilSafeCall(hipUnbindTexture(cellStartTex));
		cutilSafeCall(hipUnbindTexture(cellEndTex));
		#endif
	}

void calcAcceleration(
	float* acceleration,
	float* sortedPos,
	float* sortedReferencePos,
	float* uDisplacementGradient,
	float* vDisplacementGradient,
	float* wDisplacementGradient, 
	float* sortedVel,
	float* Measures,										
	uint* gridParticleIndex,
	uint* cellStart,
	uint* cellEnd,
	uint numParticles,
	uint numGridCells)
{
	#if USE_TEX
    cutilSafeCall(hipBindTexture(0, oldPosTex, sortedPos, numParticles*sizeof(float4)));
	cutilSafeCall(hipBindTexture(0, oldReferencePosTex, sortedReferencePos, numParticles*sizeof(float4)));
	cutilSafeCall(hipBindTexture(0, olduDisplacementGradientTex, uDisplacementGradient, numParticles*sizeof(float4)));
	cutilSafeCall(hipBindTexture(0, oldvDisplacementGradientTex, vDisplacementGradient, numParticles*sizeof(float4)));
	cutilSafeCall(hipBindTexture(0, oldwDisplacementGradientTex, wDisplacementGradient, numParticles*sizeof(float4)));
    cutilSafeCall(hipBindTexture(0, oldVelTex, sortedVel, numParticles*sizeof(float4)));
	cutilSafeCall(hipBindTexture(0, oldMeasuresTex, Measures, numParticles*sizeof(float4)));
    cutilSafeCall(hipBindTexture(0, cellStartTex, cellStart, numGridCells*sizeof(uint)));
    cutilSafeCall(hipBindTexture(0, cellEndTex, cellEnd, numGridCells*sizeof(uint)));    
	#endif

	uint numThreads, numBlocks;
    computeGridSize(numParticles, 64, numBlocks, numThreads);

    calcAccelerationD<<< numBlocks, numThreads >>>(		
	    (float4*)acceleration,
	    (float4*)sortedPos,	
		(float4*)sortedReferencePos,	
		(float4*)uDisplacementGradient,	
		(float4*)vDisplacementGradient,	
		(float4*)wDisplacementGradient,	
		(float4*)sortedVel, 
	    (float4*)Measures,		
		gridParticleIndex,
		cellStart,
		cellEnd,
		numParticles);

    cutilCheckMsg("Kernel execution failed");

	#if USE_TEX
    cutilSafeCall(hipUnbindTexture(oldPosTex));
	cutilSafeCall(hipUnbindTexture(oldReferencePosTex));
	cutilSafeCall(hipUnbindTexture(olduDisplacementGradientTex));
	cutilSafeCall(hipUnbindTexture(oldvDisplacementGradientTex));
	cutilSafeCall(hipUnbindTexture(oldwDisplacementGradientTex));
    cutilSafeCall(hipUnbindTexture(oldVelTex));
	cutilSafeCall(hipUnbindTexture(oldMeasuresTex));
    cutilSafeCall(hipUnbindTexture(cellStartTex));
    cutilSafeCall(hipUnbindTexture(cellEndTex));
	#endif
}
}// extern "C"
