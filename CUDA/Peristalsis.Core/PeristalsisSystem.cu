#include "hip/hip_runtime.h"
#include <cutil_inline.h>
#include <cstdlib>
#include <cstdio>
#include <string.h>
#include <GL/freeglut.h>
#include <cuda_gl_interop.h>

#include "thrust/device_ptr.h"
#include "thrust/for_each.h"
#include "thrust/iterator/zip_iterator.h"
#include "thrust/sort.h"

#include "peristalsisKernel.cu"
#include "peristalsisDensity.cu"
#include "peristalsisPressureForce.cu"
#include "peristalsisViscousForce.cu"
#include "peristalsisIntegrate.cu"

extern "C"
{	
	void setParameters(Peristalsiscfg *hostParams){
		cutilSafeCall( hipMemcpyToSymbol(HIP_SYMBOL(cfg), hostParams, sizeof(Peristalsiscfg)) );
	}

	uint iDivUp(uint a, uint b){
		return (a % b != 0) ? (a / b + 1) : (a / b);
	}

	void computeGridSize(uint n, uint blockSize, uint &numBlocks, uint &numThreads){
		numThreads = min(blockSize, n);
		numBlocks = iDivUp(n, numThreads);
	}

	void cudaGLInit(int argc, char **argv)
	{   
		if( cutCheckCmdLineFlag(argc, (const char**)argv, "device") ) {
			cutilDeviceInit(argc, argv);
		} else {
			cudaGLSetGLDevice( cutGetMaxGflopsDeviceId() );
		}
	}

	void registerGLBufferObject(uint vbo, struct hipGraphicsResource **cuda_vbo_resource)
	{
		cutilSafeCall(hipGraphicsGLRegisterBuffer(cuda_vbo_resource, vbo, 
							   cudaGraphicsMapFlagsNone));
	}

	void unregisterGLBufferObject(struct hipGraphicsResource *cuda_vbo_resource)
	{
		cutilSafeCall(hipGraphicsUnregisterResource(cuda_vbo_resource));	
	}

	void *mapGLBufferObject(struct hipGraphicsResource **cuda_vbo_resource)
	{
		void *ptr;
		cutilSafeCall(hipGraphicsMapResources(1, cuda_vbo_resource, 0));
		size_t num_bytes; 
		cutilSafeCall(hipGraphicsResourceGetMappedPointer((void **)&ptr, &num_bytes,  
								   *cuda_vbo_resource));
		return ptr;
	}

	void unmapGLBufferObject(struct hipGraphicsResource *cuda_vbo_resource)
	{
	   cutilSafeCall(hipGraphicsUnmapResources(1, &cuda_vbo_resource, 0));
	}

	void allocateArray(void **devPtr, size_t size)
	{
		cutilSafeCall(hipMalloc(devPtr, size));
	}

	void freeArray(void *devPtr)
	{
		cutilSafeCall(hipFree(devPtr));
	}

	void copyArrayToDevice(void* device, const void* host, int offset, int size)
	{
		cutilSafeCall(hipMemcpy((char *) device + offset, host, size, hipMemcpyHostToDevice));
	}

	void ExtConfigureBoundary(
		float* pos,
		float currentWaveHeight,
		uint numParticles){
			uint numThreads, numBlocks;
			computeGridSize(numParticles, 256, numBlocks, numThreads);

			configureBoundaryD<<< numBlocks, numThreads >>>(
				(float4*)pos,
				currentWaveHeight,
				numParticles);
		    
			cutilCheckMsg("configureBoundary kernel execution failed");
	}

	void calculatePeristalsisHash(
		uint* gridParticleHash,
		uint* gridParticleIndex,
		float* pos, 
		int numParticles){
			uint numThreads, numBlocks;
			computeGridSize(numParticles, 256, numBlocks, numThreads);

			calculatePeristalsisHashD<<< numBlocks, numThreads >>>(
				gridParticleHash,
				gridParticleIndex,
				(float4 *) pos,
				numParticles);
		    
			cutilCheckMsg("Kernel execution failed: calculatePeristalsisHashD");
	}
	
	void sortParticles(uint *dHash, uint *dIndex, uint numParticles)
	{
		thrust::sort_by_key(thrust::device_ptr<uint>(dHash),
							thrust::device_ptr<uint>(dHash + numParticles),
							thrust::device_ptr<uint>(dIndex));
	}

	void reorderPeristalsisData(
		uint*  cellStart,
		uint*  cellEnd,
		float* sortedPos,
		float* sortedVel,
		uint*  gridParticleHash,
		uint*  gridParticleIndex,
		float* oldPos,
		float* oldVel,
		uint   numParticles,
		uint   numCells){
			uint numThreads, numBlocks;
			computeGridSize(numParticles, 256, numBlocks, numThreads);

			cutilSafeCall(hipMemset(cellStart, 0xffffffff, numCells*sizeof(uint)));

			#if USE_TEX
				cutilSafeCall(hipBindTexture(0, oldPosTex, oldPos, numParticles*sizeof(float4)));
				cutilSafeCall(hipBindTexture(0, oldVelTex, oldVel, numParticles*sizeof(float4)));
			#endif

				uint smemSize = sizeof(uint)*(numThreads+1);
				reorderPeristalsisDataD<<< numBlocks, numThreads, smemSize>>>(
					cellStart,
					cellEnd,
					(float4 *) sortedPos,
					(float4 *) sortedVel,
					gridParticleHash,
					gridParticleIndex,
					(float4 *) oldPos,
					(float4 *) oldVel,
					numParticles);
				cutilCheckMsg("Kernel execution failed: reorderPeristalsisDataD");

			#if USE_TEX
				cutilSafeCall(hipUnbindTexture(oldPosTex));
				cutilSafeCall(hipUnbindTexture(oldVelTex));
			#endif
	}

	void computeDensityVariation(			
		float* measures,
		float* measuresInput,
		float* sortedPos,			
		uint* gridParticleIndex,
		uint* cellStart,
		uint* cellEnd,
		uint numParticles,
		uint numGridCells){
			#if USE_TEX
			cutilSafeCall(hipBindTexture(0, oldPosTex, sortedPos, numParticles*sizeof(float4)));
			cutilSafeCall(hipBindTexture(0, oldMeasuresTex, measuresInput, numParticles*sizeof(float4)));			
			cutilSafeCall(hipBindTexture(0, cellStartTex, cellStart, numGridCells*sizeof(uint)));
			cutilSafeCall(hipBindTexture(0, cellEndTex, cellEnd, numGridCells*sizeof(uint)));    
			#endif

			uint numThreads, numBlocks;
			computeGridSize(numParticles, 64, numBlocks, numThreads);

			computeDensityVariationD<<< numBlocks, numThreads >>>(										  
				(float4*)measures,
				(float4*)measuresInput,
				(float4*)sortedPos,  
				gridParticleIndex,
				cellStart,
				cellEnd,
				numParticles);

			cutilCheckMsg("Kernel execution failed");

			#if USE_TEX
			cutilSafeCall(hipUnbindTexture(oldPosTex));
			cutilSafeCall(hipUnbindTexture(oldMeasuresTex));	
			cutilSafeCall(hipUnbindTexture(cellStartTex));
			cutilSafeCall(hipUnbindTexture(cellEndTex));
			#endif
	}

	void computeViscousForce(
		float* viscousForce,
		float* sortedMeasures,
		float* sortedPos,			
		float* sortedVel,
		uint* gridParticleIndex,
		uint* cellStart,
		uint* cellEnd,
		uint numParticles,
		float elapsedTime,
		uint numGridCells){
			#if USE_TEX
			cutilSafeCall(hipBindTexture(0, oldPosTex, sortedPos, numParticles*sizeof(float4)));
			cutilSafeCall(hipBindTexture(0, oldVelTex, sortedVel, numParticles*sizeof(float4)));		
			cutilSafeCall(hipBindTexture(0, oldMeasuresTex, sortedMeasures, numParticles*sizeof(float4)));
			cutilSafeCall(hipBindTexture(0, cellStartTex, cellStart, numGridCells*sizeof(uint)));
			cutilSafeCall(hipBindTexture(0, cellEndTex, cellEnd, numGridCells*sizeof(uint)));    
			#endif

			uint numThreads, numBlocks;
			computeGridSize(numParticles, 64, numBlocks, numThreads);

			computeViscousForceD<<< numBlocks, numThreads >>>(
				(float4*)viscousForce,
				(float4*)sortedMeasures,		
				(float4*)sortedPos,                                          
				(float4*)sortedVel, 
				gridParticleIndex,
				cellStart,
				cellEnd,
				numParticles,
				elapsedTime);

			cutilCheckMsg("Kernel execution failed");

			#if USE_TEX
			cutilSafeCall(hipUnbindTexture(oldPosTex));
			cutilSafeCall(hipUnbindTexture(oldVelTex));	
			cutilSafeCall(hipUnbindTexture(oldMeasuresTex));
			cutilSafeCall(hipUnbindTexture(cellStartTex));
			cutilSafeCall(hipUnbindTexture(cellEndTex));
			#endif
	}

	void computePressureForce(
		float* pressureForce,		
		float* sortedMeasures,
		float* sortedPos,					
		uint* gridParticleIndex,
		uint* cellStart,
		uint* cellEnd,
		uint numParticles,
		float elapsedTime,
		uint numGridCells){
			#if USE_TEX
			cutilSafeCall(hipBindTexture(0, oldPosTex, sortedPos, numParticles*sizeof(float4)));
			cutilSafeCall(hipBindTexture(0, oldMeasuresTex, sortedMeasures, numParticles*sizeof(float4)));			
			cutilSafeCall(hipBindTexture(0, cellStartTex, cellStart, numGridCells*sizeof(uint)));
			cutilSafeCall(hipBindTexture(0, cellEndTex, cellEnd, numGridCells*sizeof(uint)));    
			#endif

			uint numThreads, numBlocks;
			computeGridSize(numParticles, 64, numBlocks, numThreads);

			computePressureForceD<<< numBlocks, numThreads >>>(
				(float4*)pressureForce,
				(float4*)sortedMeasures,										  
				(float4*)sortedPos,				
				gridParticleIndex,
				cellStart,
				cellEnd,
				numParticles,
				elapsedTime);

			cutilCheckMsg("Kernel execution failed");

			#if USE_TEX
			cutilSafeCall(hipUnbindTexture(oldPosTex));
			cutilSafeCall(hipUnbindTexture(oldMeasuresTex));			
			cutilSafeCall(hipUnbindTexture(cellStartTex));
			cutilSafeCall(hipUnbindTexture(cellEndTex));
			#endif
	}

	void predictCoordinates(
		float* predictedPosition,
		float* predictedVelocity,
		float* pos,
		float* vel,  
		float* viscousForce,
		float* pressureForce,
		uint numParticles){
			uint numThreads, numBlocks;
			computeGridSize(numParticles, 256, numBlocks, numThreads);

			predictCoordinatesD<<< numBlocks, numThreads >>>(
				(float4*)predictedPosition,
				(float4*)predictedVelocity,
				(float4*)pos,
				(float4*)vel,
				(float4*)viscousForce,
				(float4*)pressureForce,
				numParticles);
		    
			cutilCheckMsg("predictCoordinates kernel execution failed");
	}
	
	void computeCoordinates(
		float* pos,
		float* vel,  
		float* velLeapFrog,
		float* viscousForce,
		float* pressureForce,
		float elapsedTime,
		uint numParticles){
			uint numThreads, numBlocks;
			computeGridSize(numParticles, 256, numBlocks, numThreads);

			computeCoordinatesD<<< numBlocks, numThreads >>>(
				(float4*)pos,
				(float4*)vel,
				(float4*)velLeapFrog,
				(float4*)viscousForce,
				(float4*)pressureForce,
				elapsedTime,
				numParticles);
		    
			cutilCheckMsg("computeCoordinates kernel execution failed");
	}

}// extern "C"

