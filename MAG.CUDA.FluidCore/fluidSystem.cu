#include "hip/hip_runtime.h"
#include <cutil_inline.h>
#include <cstdlib>
#include <cstdio>
#include <string.h>

#if defined(__APPLE__) || defined(MACOSX)
#include <GLUT/glut.h>
#else
#include <GL/glut.h>
#endif

#include <cuda_gl_interop.h>

#include "fluid_kernel.cu"

extern "C"
{

void cudaInit(int argc, char **argv)
{   
    // use command-line specified CUDA device, otherwise use device with highest Gflops/s
    if( cutCheckCmdLineFlag(argc, (const char**)argv, "device") ) {
        cutilDeviceInit(argc, argv);
    } else {
        hipSetDevice( cutGetMaxGflopsDeviceId() );
    }
}

void cudaGLInit(int argc, char **argv)
{   
    // use command-line specified CUDA device, otherwise use device with highest Gflops/s
    if( cutCheckCmdLineFlag(argc, (const char**)argv, "device") ) {
        cutilDeviceInit(argc, argv);
    } else {
        cudaGLSetGLDevice( cutGetMaxGflopsDeviceId() );
    }
}

void allocateArray(void **devPtr, size_t size)
{
    cutilSafeCall(hipMalloc(devPtr, size));
}

void freeArray(void *devPtr)
{
    cutilSafeCall(hipFree(devPtr));
}

void threadSync()
{
    cutilSafeCall(hipDeviceSynchronize());
}

void copyArrayToDevice(void* device, const void* host, int offset, int size)
{
    cutilSafeCall(hipMemcpy((char *) device + offset, host, size, hipMemcpyHostToDevice));
}

void registerGLBufferObject(uint vbo, struct hipGraphicsResource **cuda_vbo_resource)
{
    cutilSafeCall(hipGraphicsGLRegisterBuffer(cuda_vbo_resource, vbo, 
					       cudaGraphicsMapFlagsNone));
}

void unregisterGLBufferObject(struct hipGraphicsResource *cuda_vbo_resource)
{
    cutilSafeCall(hipGraphicsUnregisterResource(cuda_vbo_resource));	
}

void *mapGLBufferObject(struct hipGraphicsResource **cuda_vbo_resource)
{
    void *ptr;
    cutilSafeCall(hipGraphicsMapResources(1, cuda_vbo_resource, 0));
    size_t num_bytes; 
    cutilSafeCall(hipGraphicsResourceGetMappedPointer((void **)&ptr, &num_bytes,  
						       *cuda_vbo_resource));
    return ptr;
}

void unmapGLBufferObject(struct hipGraphicsResource *cuda_vbo_resource)
{
   cutilSafeCall(hipGraphicsUnmapResources(1, &cuda_vbo_resource, 0));
}

void copyArrayFromDevice(void* host, const void* device, 
			 struct hipGraphicsResource **cuda_vbo_resource, int size)
{   
    if (cuda_vbo_resource)
	device = mapGLBufferObject(cuda_vbo_resource);

    cutilSafeCall(hipMemcpy(host, device, size, hipMemcpyDeviceToHost));
    
    if (cuda_vbo_resource)
	unmapGLBufferObject(*cuda_vbo_resource);
}

void setParameters(SimParams *hostParams)
{
    // copy parameters to constant memory
    cutilSafeCall( hipMemcpyToSymbol(HIP_SYMBOL(params), hostParams, sizeof(SimParams)) );
}

//Round a / b to nearest higher integer value
uint iDivUp(uint a, uint b){
    return (a % b != 0) ? (a / b + 1) : (a / b);
}

// compute grid and thread block size for a given number of elements
void computeGridSize(uint n, uint blockSize, uint &numBlocks, uint &numThreads)
{
    numThreads = min(blockSize, n);
    numBlocks = iDivUp(n, numThreads);
}

void integrateSystem(float *pos,
                     float *vel,  
					 float* velLeapFrog,
					 float *acc,
                     uint numParticles)
{
    uint numThreads, numBlocks;
    computeGridSize(numParticles, 256, numBlocks, numThreads);

    // execute the kernel
    integrate<<< numBlocks, numThreads >>>((float4*)pos,
                                           (float4*)vel,
										   (float4*)velLeapFrog,
										   (float4*)acc,
                                           numParticles);
    
    // check if kernel invocation generated an error
    cutilCheckMsg("integrate kernel execution failed");
}

void calcHash(uint*  gridParticleHash,
              uint*  gridParticleIndex,
              float* pos, 
              int    numParticles)
{
    uint numThreads, numBlocks;
    computeGridSize(numParticles, 256, numBlocks, numThreads);

    // execute the kernel
    calcHashD<<< numBlocks, numThreads >>>(gridParticleHash,
                                           gridParticleIndex,
                                           (float4 *) pos,
                                           numParticles);
    
    // check if kernel invocation generated an error
    cutilCheckMsg("Kernel execution failed");
}

void reorderDataAndFindCellStart(uint*  cellStart,
							     uint*  cellEnd,
							     float* sortedPos,
							     float* sortedVel,
                                 uint*  gridParticleHash,
                                 uint*  gridParticleIndex,
							     float* oldPos,
							     float* oldVel,
							     uint   numParticles,
							     uint   numCells)
{
    uint numThreads, numBlocks;
    computeGridSize(numParticles, 256, numBlocks, numThreads);

    // set all cells to empty
	cutilSafeCall(hipMemset(cellStart, 0xffffffff, numCells*sizeof(uint)));

	#if USE_TEX
		cutilSafeCall(hipBindTexture(0, oldPosTex, oldPos, numParticles*sizeof(float4)));
		cutilSafeCall(hipBindTexture(0, oldVelTex, oldVel, numParticles*sizeof(float4)));
	#endif

		uint smemSize = sizeof(uint)*(numThreads+1);
		reorderDataAndFindCellStartD<<< numBlocks, numThreads, smemSize>>>(
			cellStart,
			cellEnd,
			(float4 *) sortedPos,
			(float4 *) sortedVel,
			gridParticleHash,
			gridParticleIndex,
			(float4 *) oldPos,
			(float4 *) oldVel,
			numParticles);
		cutilCheckMsg("Kernel execution failed: reorderDataAndFindCellStartD");

	#if USE_TEX
		cutilSafeCall(hipUnbindTexture(oldPosTex));
		cutilSafeCall(hipUnbindTexture(oldVelTex));
	#endif
}

void calcDensityAndPressure(			
			float* measures,
			float* sortedPos,			
			uint* gridParticleIndex,
			uint* cellStart,
			uint* cellEnd,
			uint numParticles,
			uint numGridCells)
{

	#if USE_TEX
    cutilSafeCall(hipBindTexture(0, oldPosTex, sortedPos, numParticles*sizeof(float4)));
    cutilSafeCall(hipBindTexture(0, cellStartTex, cellStart, numGridCells*sizeof(uint)));
    cutilSafeCall(hipBindTexture(0, cellEndTex, cellEnd, numGridCells*sizeof(uint)));    
	#endif

	uint numThreads, numBlocks;
    computeGridSize(numParticles, 64, numBlocks, numThreads);

    // execute the kernel
    calcDensityAndPressureD<<< numBlocks, numThreads >>>(
										  /*(float*)density,
										  (float*)pressure,*/
										  (float4*)measures,
                                          (float4*)sortedPos,                                          
                                          gridParticleIndex,
                                          cellStart,
                                          cellEnd,
                                          numParticles);

    // check if kernel invocation generated an error
    cutilCheckMsg("Kernel execution failed");

	#if USE_TEX
    cutilSafeCall(hipUnbindTexture(oldPosTex));
    cutilSafeCall(hipUnbindTexture(cellStartTex));
    cutilSafeCall(hipUnbindTexture(cellEndTex));
	#endif
}

void calcAndApplyAcceleration(
			float* acceleration,
			float* sortedMeasures,			
			float* sortedPos,			
			float* sortedVel,
			uint* gridParticleIndex,
			uint* cellStart,
			uint* cellEnd,
			uint numParticles,
			uint numGridCells)
{
	#if USE_TEX
    cutilSafeCall(hipBindTexture(0, oldPosTex, sortedPos, numParticles*sizeof(float4)));
    cutilSafeCall(hipBindTexture(0, oldVelTex, sortedVel, numParticles*sizeof(float4)));
	cutilSafeCall(hipBindTexture(0, oldMeasuresTex, sortedMeasures, numParticles*sizeof(float4)));
    cutilSafeCall(hipBindTexture(0, cellStartTex, cellStart, numGridCells*sizeof(uint)));
    cutilSafeCall(hipBindTexture(0, cellEndTex, cellEnd, numGridCells*sizeof(uint)));    
	#endif

	uint numThreads, numBlocks;
    computeGridSize(numParticles, 64, numBlocks, numThreads);

    // execute the kernel
    calcAndApplyAccelerationD<<< numBlocks, numThreads >>>(
										  (float4*)acceleration,
										  (float4*)sortedMeasures,
										  /*(float*)density,
										  (float*)pressure,*/
                                          (float4*)sortedPos,                                          
										  (float4*)sortedVel, 
                                          gridParticleIndex,
                                          cellStart,
                                          cellEnd,
                                          numParticles);

    // check if kernel invocation generated an error
    cutilCheckMsg("Kernel execution failed");

	#if USE_TEX
    cutilSafeCall(hipUnbindTexture(oldPosTex));
    cutilSafeCall(hipUnbindTexture(oldVelTex));
	cutilSafeCall(hipUnbindTexture(oldMeasuresTex));
    cutilSafeCall(hipUnbindTexture(cellStartTex));
    cutilSafeCall(hipUnbindTexture(cellEndTex));
	#endif
}
}   // extern "C"
