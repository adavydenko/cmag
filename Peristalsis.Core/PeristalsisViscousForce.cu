#include "hip/hip_runtime.h"
#include "cutil_math.h"
#include "peristalsisUtil.cu"

//__device__ struct BottomF {	
//	float x0, y0, t;
//	float A, B, Wx, Wy;
//
//	__device__ BottomF(){
//		A = cfg.amplitude;
//		B = cfg.BoundaryHeight();
//		Wx = cfg.worldOrigin.x;
//		Wy = cfg.worldOrigin.y;
//	}
//
//	__device__ float operator() (const float x) {	
//		return x0 - x + (y0 - Wy - B - A + A * sinf(-cfg.sigma * (x - Wx) + cfg.wave_speed * t)) *
//			A * cosf(-cfg.sigma * (x - Wx) + cfg.wave_speed * t) * cfg.sigma;						
//	}
//
//	__device__ float df(const float x) {		
//		return -1 - powf(A * cosf(-cfg.sigma * (x - Wx) + cfg.wave_speed * t) * cfg.sigma,2) +
//			(y0 - Wy - B - A + A * sinf(-cfg.sigma * (x - Wx) + cfg.wave_speed * t)) *
//			A * sinf(-cfg.sigma * (x - Wx) + cfg.sigma * t) * powf(cfg.sigma, 2);
//	}
//};
//
//__device__ struct TopF {	
//	float x0, y0, t;
//	float A, B, Wx, Wy, F;
//
//	__device__ TopF(){
//		A = cfg.amplitude;
//		B = cfg.BoundaryHeight();
//		Wx = cfg.worldOrigin.x;
//		Wy = cfg.worldOrigin.y;
//		F = cfg.FluidHeight();
//	}
//
//	__device__ float operator() (const float x) {	
//		return x0 - x + (y0 - Wy - B - A - F - A * sinf(-cfg.sigma * (x - Wx) + cfg.wave_speed * t)) *
//			A * cosf(-cfg.sigma * (x - Wx) + cfg.wave_speed * t) * cfg.sigma;						
//	}
//
//	__device__ float df(const float x) {		
//		return -1 - powf(A * cosf(-cfg.sigma * (x - Wx) + cfg.wave_speed * t) * cfg.sigma,2) -
//			(y0 - Wy - B - A - F - A * sinf(-cfg.sigma * (x - Wx) + cfg.wave_speed * t)) *
//			A * sinf(-cfg.sigma * (x - Wx) + cfg.sigma * t) * powf(cfg.sigma, 2);
//	}
//};
//
//template <class T>
//__device__ float rtnewt(T &funcd, const float x1, const float x2, const float xacc) {
//	const int JMAX=20;
//	float rtn=0.5*(x1+x2);
//	for (int j=0;j<JMAX;j++) {
//		float f=funcd(rtn);
//		float df=funcd.df(rtn);
//		float dx=f/df;
//		rtn -= dx;
//		if ((x1-rtn)*(rtn-x2) < 0.0)
//			return 0;//-1;
//		if (abs(dx) < xacc) return rtn;
//	}
//	return 0;//-1;
//}


__device__ float4 getVelocityDiff(
	float4 iVelocity, 
	float4 iPosition, 
	float4 jVelocity,
	float4 jPosition,
	float elapsedTime)
{		
	/*float bottomBoundary = cfg.worldOrigin.y + cfg.BoundaryHeight() + cfg.amplitude;	
	float topBoundary = bottomBoundary + cfg.fluidParticlesSize.y * 2.0f * cfg.radius;		
	if(jPosition.w < 0.0f)
	{
		float distanceA = topBoundary - iPosition.y;
		float distanceB = jPosition.y - topBoundary;
		float beta = fmin(1000.0f, 1 + distanceB / distanceA);
		return beta * iVelocity;
	}
	
	if(jPosition.w > 0.0f)
	{
		float distanceA = iPosition.y - bottomBoundary;
		float distanceB = bottomBoundary - jPosition.y;
		float beta = fmin(1000.0f, 1 + distanceB / distanceA);
		return beta * iVelocity;
	}*/

	/*float A = cfg.amplitude;
	float B = cfg.BoundaryHeight();
	float Wx = cfg.worldOrigin.x;
	float Wy = cfg.worldOrigin.y;
	float F = cfg.FluidHeight();*/

	//if(jPosition.w < 0.0f)//top
	//{
	//	TopF fx;
	//	fx.x0 = iPosition.x;
	//	fx.y0 = iPosition.y;
	//	fx.t = elapsedTime;
	//	float xA = rtnewt(fx, cfg.worldOrigin.x, -cfg.worldOrigin.x, cfg.radius / 100);		
	//	float yA = Wy + B + A + F + A * sinf(-cfg.sigma * (xA - Wx) + cfg.wave_speed * elapsedTime);
	//	float distA = sqrtf(powf(iPosition.x - xA,2) + powf(iPosition.y - yA,2));	
	//	float k = -A * cosf(-cfg.sigma * (xA - Wx) + cfg.wave_speed * elapsedTime) * cfg.sigma;

	//	float AA = -k;
	//	float BB = 1;
	//	float CC = k * xA - yA;
	//	float distB = abs(AA* jPosition.x + BB * jPosition.y + CC) / sqrt(AA * AA + 1);

	//	float beta = fmin(1.5f, 1 + distB / distA);
	//	return beta * (iVelocity); 
	//}
	//
	//
	//if(jPosition.w > 0.0f)//bottom
	//{
	//	BottomF fx;
	//	fx.x0 = iPosition.x;
	//	fx.y0 = iPosition.y;
	//	fx.t = elapsedTime;
	//	float xA = rtnewt(fx, cfg.worldOrigin.x, -cfg.worldOrigin.x, cfg.radius / 100);		
	//	float yA = Wy + B + A - A * sinf(-cfg.sigma * (xA - Wx) + cfg.wave_speed * elapsedTime);
	//	float distA = sqrtf(powf(iPosition.x - xA,2) + powf(iPosition.y - yA,2));	
	//	float k = A * cosf(-cfg.sigma * (xA - Wx) + cfg.wave_speed * elapsedTime) * cfg.sigma;

	//	float AA = -k;
	//	float BB = 1;
	//	float CC = k * xA - yA;
	//	float distB = abs(AA* jPosition.x + BB * jPosition.y + CC) / sqrt(AA * AA + 1);

	//	float beta = fmin(1.5f, 1 + distB / distA);
	//	return beta * (iVelocity); 
	//}
	
	return iVelocity - jVelocity;	
}

__device__ float3 sumViscosity(
	int3    gridPos,
	uint    index,
	float4  pos,
	float4* oldPos, 
	float4  vel,
	float4* oldVel,
	float density,
	float pressure,
	float4* oldMeasures,
	uint*   cellStart,
	uint*   cellEnd,
	float elapsedTime){
		uint gridHash = calcGridHash(gridPos);
		int3 shift = make_int3(EvaluateShift(gridPos.x, cfg.gridSize.x),
			EvaluateShift(gridPos.y, cfg.gridSize.y),
			EvaluateShift(gridPos.z, cfg.gridSize.z));							

		uint startIndex = FETCH(cellStart, gridHash);	    
		float3 force = make_float3(0.0f);
		if (startIndex != 0xffffffff) {               
			uint endIndex = FETCH(cellEnd, gridHash);
			for(uint j=startIndex; j<endIndex; j++) {
				if (j != index) {             
					float4 pos2 = FETCH(oldPos, j);
					float4 vel2 = FETCH(oldVel, j);
					float4 measure = FETCH(oldMeasures, j);
					float density2 = measure.x;
					float pressure2 = measure.y;

					float3 relPos = make_float3(pos.x - (pos2.x + shift.x * cfg.worldSize.x),
						pos.y - (pos2.y + shift.y * cfg.worldSize.y),
						pos.z - (pos2.z + shift.z * cfg.worldSize.z));
					  										
					float dist = length(relPos);
					float q = dist / cfg.smoothingRadius;									

					float coeff = 7.0f / (2 * HIP_PI_F * powf(cfg.smoothingRadius, 3));
					float4 Vab = getVelocityDiff(vel, pos, vel2, pos2, elapsedTime);
					if(q < 2){

						float temp = coeff * (-pow(1 - 0.5f * q,3) * (2 * q + 1) + pow(1 - 0.5f * q, 4));
					/*	force += cfg.particleMass * temp * (-1.0f *
							(pressure / powf(density,2) + pressure2 / powf(density2,2)) * 
							normalize(relPos) + (cfg.mu + cfg.mu) * 
							make_float3(Vab) / (density * density2 * dist));	*/
						force += cfg.particleMass * temp * (cfg.mu + cfg.mu) * 
							make_float3(Vab) / (density * density2 * dist);
					}
				}
			}
		}
		return force;				
}

__global__ void computeViscousForceD(
	float4* viscousForce,	
	float4* oldMeasures,
	float4* oldPos,			
	float4* oldVel,
	uint* gridParticleIndex,
	uint* cellStart,
	uint* cellEnd,
	uint numParticles,
	float elapsedTime){
		uint index = __mul24(blockIdx.x,blockDim.x) + threadIdx.x;
		if (index >= numParticles) return;    

		float4 pos = FETCH(oldPos, index);
		float4 vel = FETCH(oldVel, index);
		float4 measure = FETCH(oldMeasures,index);
		float density = measure.x;
		float pressure = measure.y;

		int3 gridPos = calcGridPos(make_float3(pos));

		float3 force = make_float3(0.0f);		
		for(int z=-cfg.cellcount; z<=cfg.cellcount; z++) {
			for(int y=-cfg.cellcount; y<=cfg.cellcount; y++) {
				for(int x=-cfg.cellcount; x<=cfg.cellcount; x++) {
					int3 neighbourPos = gridPos + make_int3(x, y, z);
					force += sumViscosity(
						neighbourPos, 
						index, 
						pos, 
						oldPos,
						vel,
						oldVel,
						density,
						pressure,	
						oldMeasures,
						cellStart, 
						cellEnd,
						elapsedTime);
				}
			}
		}
		uint originalIndex = gridParticleIndex[index];							
		viscousForce[originalIndex] = make_float4(force, 0.0f);
}